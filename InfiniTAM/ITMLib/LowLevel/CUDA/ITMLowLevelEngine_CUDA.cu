#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMLowLevelEngine_CUDA.h"

#include "../Shared/ITMLowLevelEngine_Shared.h"
#include "../../Utils/ITMCUDAUtils.h"
#include "../../Utils/ITMLibDefines.h"
#include "../../../ORUtils/CUDADefines.h"

using namespace ITMLib;

ITMLowLevelEngine_CUDA::ITMLowLevelEngine_CUDA(void)
{
	ORcudaSafeCall(hipMalloc((void**)&counterTempData_device, sizeof(int)));
	ORcudaSafeCall(hipHostMalloc((void**)&counterTempData_host, sizeof(int)));
}

ITMLowLevelEngine_CUDA::~ITMLowLevelEngine_CUDA(void)
{
	ORcudaSafeCall(hipFree(counterTempData_device));
	ORcudaSafeCall(hipHostFree(counterTempData_host));
}

__global__ void filterSubsample_device(Vector4u *imageData_out, Vector2i newDims, const Vector4u *imageData_in, Vector2i oldDims);

__global__ void filterSubsampleWithHoles_device(float *imageData_out, Vector2i newDims, const float *imageData_in, Vector2i oldDims);
__global__ void filterSubsampleWithHoles_device(Vector4f *imageData_out, Vector2i newDims, const Vector4f *imageData_in, Vector2i oldDims);

__global__ void gradientX_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize);
__global__ void gradientY_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize);

__global__ void countValidDepths_device(const float *imageData_in, int imgSizeTotal, int *counterTempData_device);

// host methods

void ITMLowLevelEngine_CUDA::CopyImage(ITMUChar4Image *image_out, const ITMUChar4Image *image_in) const
{
	Vector4u *dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u *src = image_in->GetData(MEMORYDEVICE_CUDA);

	ORcudaSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(Vector4u), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::CopyImage(ITMFloatImage *image_out, const ITMFloatImage *image_in) const
{
	float *dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const float *src = image_in->GetData(MEMORYDEVICE_CUDA);

	ORcudaSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(float), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::CopyImage(ITMFloat4Image *image_out, const ITMFloat4Image *image_in) const
{
	Vector4f *dest = image_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4f *src = image_in->GetData(MEMORYDEVICE_CUDA);

	ORcudaSafeCall(hipMemcpy(dest, src, image_in->dataSize * sizeof(Vector4f), hipMemcpyDeviceToDevice));
}

void ITMLowLevelEngine_CUDA::FilterSubsample(ITMUChar4Image *image_out, const ITMUChar4Image *image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims; newDims.x = image_in->noDims.x / 2; newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const Vector4u *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	Vector4u *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	filterSubsample_device << <gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
}

void ITMLowLevelEngine_CUDA::FilterSubsampleWithHoles(ITMFloatImage *image_out, const ITMFloatImage *image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims; newDims.x = image_in->noDims.x / 2; newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	filterSubsampleWithHoles_device << <gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
}

void ITMLowLevelEngine_CUDA::FilterSubsampleWithHoles(ITMFloat4Image *image_out, const ITMFloat4Image *image_in) const
{
	Vector2i oldDims = image_in->noDims;
	Vector2i newDims; newDims.x = image_in->noDims.x / 2; newDims.y = image_in->noDims.y / 2;

	image_out->ChangeDims(newDims);

	const Vector4f *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	Vector4f *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	filterSubsampleWithHoles_device << <gridSize, blockSize >> >(imageData_out, newDims, imageData_in, oldDims);
}

void ITMLowLevelEngine_CUDA::GradientX(ITMShort4Image *grad_out, const ITMUChar4Image *image_in) const
{
	grad_out->ChangeDims(image_in->noDims);
	Vector2i imgSize = image_in->noDims;

	Vector4s *grad = grad_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u *image = image_in->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	ORcudaSafeCall(hipMemset(grad, 0, imgSize.x * imgSize.y * sizeof(Vector4s)));

	gradientX_device << <gridSize, blockSize >> >(grad, image, imgSize);
}

void ITMLowLevelEngine_CUDA::GradientY(ITMShort4Image *grad_out, const ITMUChar4Image *image_in) const
{
	grad_out->ChangeDims(image_in->noDims);
	Vector2i imgSize = image_in->noDims;

	Vector4s *grad = grad_out->GetData(MEMORYDEVICE_CUDA);
	const Vector4u *image = image_in->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	ORcudaSafeCall(hipMemset(grad, 0, imgSize.x * imgSize.y * sizeof(Vector4s)));

	gradientY_device << <gridSize, blockSize >> >(grad, image, imgSize);
}

int ITMLowLevelEngine_CUDA::CountValidDepths(const ITMFloatImage *image_in) const
{
	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	Vector2i imgSize = image_in->noDims;

	dim3 blockSize(256);
	dim3 gridSize((int)ceil((float)imgSize.x*imgSize.y / (float)blockSize.x));

	ORcudaSafeCall(hipMemset(counterTempData_device, 0, sizeof(int)));
	countValidDepths_device <<<gridSize, blockSize>>>(imageData_in, imgSize.x*imgSize.y, counterTempData_device);
	ORcudaSafeCall(hipMemcpy(counterTempData_host, counterTempData_device, sizeof(int), hipMemcpyDeviceToHost));

	return *counterTempData_host;
}

// device functions

__global__ void filterSubsample_device(Vector4u *imageData_out, Vector2i newDims, const Vector4u *imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsample(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void filterSubsampleWithHoles_device(float *imageData_out, Vector2i newDims, const float *imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsampleWithHoles(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void filterSubsampleWithHoles_device(Vector4f *imageData_out, Vector2i newDims, const Vector4f *imageData_in, Vector2i oldDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsampleWithHoles(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void gradientX_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 2 || x > imgSize.x - 2 || y < 2 || y > imgSize.y - 2) return;

	gradientX(grad, x, y, image, imgSize);
}

__global__ void gradientY_device(Vector4s *grad, const Vector4u *image, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 2 || x > imgSize.x - 2 || y < 2 || y > imgSize.y - 2) return;

	gradientY(grad, x, y, image, imgSize);
}

__global__ void countValidDepths_device(const float *imageData_in, int imgSizeTotal, int *counterTempData_device)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int locId_local = threadIdx.x;

	__shared__ int dim_shared[256];
	//__shared__ bool should_prefix;

	//should_prefix = false;
	//__syncthreads();

	bool isValidPoint = false;

	if (i < imgSizeTotal)
	{
		if (imageData_in[i] > 0.0f) isValidPoint = true;
	}

	//__syncthreads();
	//if (!should_prefix) return;

	dim_shared[locId_local] = isValidPoint;
	__syncthreads();

	if (locId_local < 128) dim_shared[locId_local] += dim_shared[locId_local + 128];
	__syncthreads();
	if (locId_local < 64) dim_shared[locId_local] += dim_shared[locId_local + 64];
	__syncthreads();

	if (locId_local < 32) warpReduce(dim_shared, locId_local);

	if (locId_local == 0) atomicAdd(counterTempData_device, dim_shared[locId_local]);
}

