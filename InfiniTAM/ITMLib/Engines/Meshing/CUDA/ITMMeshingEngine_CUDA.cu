#include "hip/hip_runtime.h"
// Copyright 2016 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMMeshingEngine_CUDA.h"

using namespace ITMLib;

__global__ void findAllocateBlocks(Vector4s *visibleBlockGlobalPos, const ITMHashEntry *hashTable, int noTotalEntries)
{
	int entryId = threadIdx.x + blockIdx.x * blockDim.x;
	if (entryId > noTotalEntries - 1) return;

	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	if (currentHashEntry.ptr >= 0) 
		visibleBlockGlobalPos[currentHashEntry.ptr] = Vector4s(currentHashEntry.pos.x, currentHashEntry.pos.y, currentHashEntry.pos.z, 1);
}
