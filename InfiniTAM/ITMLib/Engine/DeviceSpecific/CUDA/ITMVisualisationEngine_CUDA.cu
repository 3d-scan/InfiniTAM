#include "hip/hip_runtime.h"
// Copyright 2014 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMVisualisationEngine_CUDA.h"
#include "ITMCUDAUtils.h"
#include "../../DeviceAgnostic/ITMRepresentationAccess.h"
#include "../../DeviceAgnostic/ITMVisualisationEngine.h"
#include "../../DeviceAgnostic/ITMSceneReconstructionEngine.h"

#include "../../../Objects/ITMRenderState_VH.h"

using namespace ITMLib::Engine;

inline dim3 getGridSize(dim3 taskSize, dim3 blockSize)
{
	return dim3((taskSize.x + blockSize.x - 1) / blockSize.x, (taskSize.y + blockSize.y - 1) / blockSize.y, (taskSize.z + blockSize.z - 1) / blockSize.z);
}

inline dim3 getGridSize(Vector2i taskSize, dim3 blockSize) { return getGridSize(dim3(taskSize.x, taskSize.y), blockSize); }

// declaration of device functions

__global__ void buildVisibleList_device(const ITMHashEntry *hashTable, /*ITMHashCacheState *cacheStates, bool useSwapping,*/ int noTotalEntries,
	int *visibleEntryIDs, int *noVisibleEntries, uchar *entriesVisibleType, Matrix4f M, Vector4f projParams, Vector2i imgSize, float voxelSize);

template<typename T>
__global__ void memsetKernel_device(T *devPtr, const T val, size_t nwords);

__global__ void projectAndSplitBlocks_device(const ITMHashEntry *hashEntries, const int *visibleEntryIDs, int noVisibleEntries,
	const Matrix4f pose_M, const Vector4f intrinsics, const Vector2i imgSize, float voxelSize, RenderingBlock *renderingBlocks,
	uint *noTotalBlocks);

__global__ void fillBlocks_device(const uint *noTotalBlocks, const RenderingBlock *renderingBlocks,
	Vector2i imgSize, Vector2f *minmaxData);

template<class TVoxel, class TIndex>
__global__ void genericRaycast_device(Vector4f *out_ptsRay, const TVoxel *voxelData, const typename TIndex::IndexData *voxelIndex,
	Vector2i imgSize, Matrix4f invM, Vector4f projParams, float oneOverVoxelSize, const Vector2f *minmaxdata, float mu);

template<class TVoxel, class TIndex>
__global__ void renderICP_device(Vector4u *outRendering, Vector4f *pointsMap, Vector4f *normalsMap, const Vector4f *ptsRay,
	const TVoxel *voxelData, const typename TIndex::IndexData *voxelIndex, float voxelSize, Vector2i imgSize, Vector3f lightSource);

template<class TVoxel, class TIndex>
__global__ void renderGrey_device(Vector4u *outRendering, const Vector4f *ptsRay, const TVoxel *voxelData,
	const typename TIndex::IndexData *voxelIndex, Vector2i imgSize, Vector3f lightSource);

template<class TVoxel, class TIndex>
__global__ void renderPointCloud_device(Vector4u *outRendering, Vector4f *locations, Vector4f *colours, uint *noTotalPoints,
	const Vector4f *ptsRay, const TVoxel *voxelData, const typename TIndex::IndexData *voxelIndex, bool skipPoints,
	float voxelSize, Vector2i imgSize, Vector3f lightSource);

template<class TVoxel, class TIndex>
__global__ void renderColour_device(Vector4u *outRendering, const Vector4f *ptsRay, const TVoxel *voxelData,
	const typename TIndex::IndexData *voxelIndex, Vector2i imgSize, Vector3f lightSource);

// class implementation

template<class TVoxel, class TIndex>
ITMVisualisationEngine_CUDA<TVoxel, TIndex>::ITMVisualisationEngine_CUDA(void)
{
	ITMSafeCall(hipMalloc((void**)&noTotalPoints_device, sizeof(uint)));
}

template<class TVoxel, class TIndex>
ITMVisualisationEngine_CUDA<TVoxel, TIndex>::~ITMVisualisationEngine_CUDA(void)
{
	ITMSafeCall(hipFree(noTotalPoints_device));
}

template<class TVoxel>
ITMVisualisationEngine_CUDA<TVoxel, ITMVoxelBlockHash>::ITMVisualisationEngine_CUDA(void)
{
	ITMSafeCall(hipMalloc((void**)&renderingBlockList_device, sizeof(RenderingBlock) * MAX_RENDERING_BLOCKS));
	ITMSafeCall(hipMalloc((void**)&noTotalBlocks_device, sizeof(uint)));
	ITMSafeCall(hipMalloc((void**)&noTotalPoints_device, sizeof(uint)));
	ITMSafeCall(hipMalloc((void**)&noVisibleEntries_device, sizeof(uint)));
}

template<class TVoxel>
ITMVisualisationEngine_CUDA<TVoxel, ITMVoxelBlockHash>::~ITMVisualisationEngine_CUDA(void)
{
	ITMSafeCall(hipFree(noTotalPoints_device));
	ITMSafeCall(hipFree(noTotalBlocks_device));
	ITMSafeCall(hipFree(renderingBlockList_device));
	ITMSafeCall(hipFree(noVisibleEntries_device));
}

template<class TVoxel, class TIndex>
ITMRenderState* ITMVisualisationEngine_CUDA<TVoxel, TIndex>::CreateRenderState(const ITMScene<TVoxel, TIndex> *scene, const Vector2i & imgSize)
{
	return new ITMRenderState(imgSize, scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max, MEMORYDEVICE_CUDA);
}

template<class TVoxel>
ITMRenderState* ITMVisualisationEngine_CUDA<TVoxel, ITMVoxelBlockHash>::CreateRenderState(const ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const Vector2i & imgSize)
{
	return new ITMRenderState_VH(ITMHashTable::noTotalEntries, imgSize, scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max, MEMORYDEVICE_CUDA);
}

template<class TVoxel, class TIndex>
void ITMVisualisationEngine_CUDA<TVoxel, TIndex>::FindVisibleBlocks(const ITMScene<TVoxel, TIndex> *scene, const ITMPose *pose,
	const ITMIntrinsics *intrinsics, ITMRenderState *renderState)
{
}

template<class TVoxel>
void ITMVisualisationEngine_CUDA<TVoxel, ITMVoxelBlockHash>::FindVisibleBlocks(const ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMPose *pose,
	const ITMIntrinsics *intrinsics, ITMRenderState *renderState)
{
	const ITMHashEntry *hashTable = scene->index.GetEntries();
	int noTotalEntries = scene->index.noVoxelBlocks;
	float voxelSize = scene->sceneParams->voxelSize;
	Vector2i imgSize = renderState->renderingRangeImage->noDims;

	Matrix4f M = pose->M;
	Vector4f projParams = intrinsics->projectionParamsSimple.all;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	ITMSafeCall(hipMemset(noVisibleEntries_device, 0, sizeof(int)));

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));
	buildVisibleList_device << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, /*cacheStates, scene->useSwapping,*/ noTotalEntries, 
		renderState_vh->GetVisibleEntryIDs(), noVisibleEntries_device, renderState_vh->GetEntriesVisibleType(), M, projParams, 
		imgSize, voxelSize);

	/*	if (scene->useSwapping)
			{
			reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries,
			noAllocatedVoxelEntries_device, entriesVisibleType);
			}*/

	ITMSafeCall(hipMemcpy(&renderState_vh->noVisibleEntries, noVisibleEntries_device, sizeof(int), hipMemcpyDeviceToHost));
}

template<class TVoxel, class TIndex>
void ITMVisualisationEngine_CUDA<TVoxel, TIndex>::CreateExpectedDepths(const ITMScene<TVoxel, TIndex> *scene, const ITMPose *pose,
	const ITMIntrinsics *intrinsics, ITMRenderState *renderState)
{
	Vector2f *minmaxData = renderState->renderingRangeImage->GetData(MEMORYDEVICE_CUDA);

	{
		dim3 blockSize(256);
		dim3 gridSize((int)ceil((float)renderState->renderingRangeImage->dataSize / (float)blockSize.x));
		Vector2f init;
		//TODO : this could be improved a bit...
		init.x = 0.2f; init.y = 3.0f;
		memsetKernel_device<Vector2f> << <gridSize, blockSize >> >(minmaxData, init, renderState->renderingRangeImage->dataSize);
	}
}

template<class TVoxel>
void ITMVisualisationEngine_CUDA<TVoxel, ITMVoxelBlockHash>::CreateExpectedDepths(const ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMPose *pose,
	const ITMIntrinsics *intrinsics, ITMRenderState *renderState)
{
	float voxelSize = scene->sceneParams->voxelSize;

	Vector2i imgSize = renderState->renderingRangeImage->noDims;
	Vector2f *minmaxData = renderState->renderingRangeImage->GetData(MEMORYDEVICE_CUDA);

	{
		dim3 blockSize(256);
		dim3 gridSize((int)ceil((float)renderState->renderingRangeImage->dataSize / (float)blockSize.x));
		Vector2f init;
		init.x = FAR_AWAY; init.y = VERY_CLOSE;
		memsetKernel_device<Vector2f> << <gridSize, blockSize >> >(minmaxData, init, renderState->renderingRangeImage->dataSize);
	}

	ITMRenderState_VH* renderState_vh = (ITMRenderState_VH*)renderState;

	//go through list of visible 8x8x8 blocks
	{
		const ITMHashEntry *hash_entries = scene->index.GetEntries();
		const int *visibleEntryIDs = renderState_vh->GetVisibleEntryIDs();
		int noVisibleEntries = renderState_vh->noVisibleEntries;

		dim3 blockSize(256);
		dim3 gridSize((int)ceil((float)noVisibleEntries / (float)blockSize.x));
		ITMSafeCall(hipMemset(noTotalBlocks_device, 0, sizeof(uint)));
		projectAndSplitBlocks_device << <gridSize, blockSize >> >(hash_entries, visibleEntryIDs, noVisibleEntries, pose->M,
			intrinsics->projectionParamsSimple.all, imgSize, voxelSize, renderingBlockList_device, noTotalBlocks_device);
	}

	uint noTotalBlocks;
	ITMSafeCall(hipMemcpy(&noTotalBlocks, noTotalBlocks_device, sizeof(uint), hipMemcpyDeviceToHost));
	if (noTotalBlocks > (unsigned)MAX_RENDERING_BLOCKS) noTotalBlocks = MAX_RENDERING_BLOCKS;

	// go through rendering blocks
	{
		// fill minmaxData
		dim3 blockSize(16, 16);
		dim3 gridSize((unsigned int)ceil((float)noTotalBlocks / 4.0f), 4);
		fillBlocks_device << <gridSize, blockSize >> >(noTotalBlocks_device, renderingBlockList_device, imgSize, minmaxData);
	}
}

template <class TVoxel, class TIndex>
static void GenericRaycast(const ITMScene<TVoxel, TIndex> *scene, const Vector2i& imgSize, const Matrix4f& invM, Vector4f projParams, const ITMRenderState *renderState, dim3 cudaBlockSize, dim3 gridSize)
{
	float voxelSize = scene->sceneParams->voxelSize;
	float oneOverVoxelSize = 1.0f / voxelSize;

	projParams.x = 1.0f / projParams.x;
	projParams.y = 1.0f / projParams.y;

	genericRaycast_device<TVoxel, TIndex> <<<gridSize, cudaBlockSize>>>(
		renderState->raycastResult->GetData(MEMORYDEVICE_CUDA),
		scene->localVBA.GetVoxelBlocks(),
		scene->index.getIndexData(),
		imgSize,
		invM,
		projParams,
		oneOverVoxelSize,
		renderState->renderingRangeImage->GetData(MEMORYDEVICE_CUDA),
		scene->sceneParams->mu
	);
}

template<class TVoxel, class TIndex>
static void FindSurface_common(const ITMScene<TVoxel, TIndex> *scene, const ITMPose *pose, const ITMIntrinsics *intrinsics, const ITMRenderState *renderState)
{
	Vector2i imgSize = renderState->raycastResult->noDims;
	dim3 cudaBlockSize(8, 8);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
	GenericRaycast(scene, imgSize, pose->invM, intrinsics->projectionParamsSimple.all, renderState, cudaBlockSize, gridSize);
}

template<class TVoxel, class TIndex>
static void RenderImage_common(const ITMScene<TVoxel, TIndex> *scene, const ITMPose *pose, const ITMIntrinsics *intrinsics, const ITMRenderState *renderState,
	ITMUChar4Image *outputImage, bool useColour)
{
	Vector2i imgSize = outputImage->noDims;
	dim3 cudaBlockSize(8, 8);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
	GenericRaycast(scene, imgSize, pose->invM, intrinsics->projectionParamsSimple.all, renderState, cudaBlockSize, gridSize);

	Vector4u *outRendering = outputImage->GetData(MEMORYDEVICE_CUDA);
	Vector4f *pointsRay = renderState->raycastResult->GetData(MEMORYDEVICE_CUDA);
	Vector3f lightSource = ComputeLightSource(pose->invM);

	if (useColour && TVoxel::hasColorInformation)
		renderColour_device<TVoxel, TIndex> <<<gridSize, cudaBlockSize>>>(outRendering, pointsRay, scene->localVBA.GetVoxelBlocks(),
		scene->index.getIndexData(), imgSize, lightSource);
	else
		renderGrey_device<TVoxel, TIndex> <<<gridSize, cudaBlockSize>>>(outRendering, pointsRay, scene->localVBA.GetVoxelBlocks(),
		scene->index.getIndexData(), imgSize, lightSource);
}

template<class TVoxel, class TIndex>
static void CreatePointCloud_common(const ITMScene<TVoxel, TIndex> *scene, const ITMView *view, ITMTrackingState *trackingState, ITMRenderState *renderState,
	bool skipPoints, uint *noTotalPoints_device)
{
	Vector2i imgSize = view->rgb->noDims;
	Matrix4f invM = trackingState->pose_d->invM * view->calib->trafo_rgb_to_depth.calib;
	dim3 cudaBlockSize(16, 16);
	dim3 gridSize = getGridSize(imgSize, cudaBlockSize);
	GenericRaycast(scene, imgSize, invM, view->calib->intrinsics_rgb.projectionParamsSimple.all, renderState, cudaBlockSize, gridSize);

	ITMSafeCall(hipMemset(noTotalPoints_device, 0, sizeof(uint)));

	Vector4f *locations = trackingState->pointCloud->locations->GetData(MEMORYDEVICE_CUDA);
	Vector4f *colours = trackingState->pointCloud->colours->GetData(MEMORYDEVICE_CUDA);
	Vector4u *outRendering = renderState->raycastImage->GetData(MEMORYDEVICE_CUDA);
	Vector4f *pointsRay = renderState->raycastResult->GetData(MEMORYDEVICE_CUDA);
	Vector3f lightSource = ComputeLightSource(invM);

	renderPointCloud_device<TVoxel, TIndex> << <gridSize, cudaBlockSize >> >(outRendering, locations, colours, noTotalPoints_device,
		pointsRay, scene->localVBA.GetVoxelBlocks(), scene->index.getIndexData(), skipPoints, scene->sceneParams->voxelSize, imgSize, lightSource);

	ITMSafeCall(hipMemcpy(&trackingState->pointCloud->noTotalPoints, noTotalPoints_device, sizeof(uint), hipMemcpyDeviceToHost));
}

template<class TVoxel, class TIndex>
void CreateICPMaps_common(const ITMScene<TVoxel, TIndex> *scene, const ITMView *view, ITMTrackingState *trackingState, ITMRenderState *renderState)
{
	Vector2i imgSize = view->depth->noDims;
	Matrix4f invM = trackingState->pose_d->invM;

	{
		dim3 cudaBlockSize(16, 16);
		dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
		GenericRaycast(scene, imgSize, invM, view->calib->intrinsics_d.projectionParamsSimple.all, renderState, cudaBlockSize, gridSize);
	}

	{
		Vector4f *pointsMap = trackingState->pointCloud->locations->GetData(MEMORYDEVICE_CUDA);
		Vector4f *normalsMap = trackingState->pointCloud->colours->GetData(MEMORYDEVICE_CUDA);
		Vector4u *outRendering = renderState->raycastImage->GetData(MEMORYDEVICE_CUDA);
		Vector4f *pointsRay = renderState->raycastResult->GetData(MEMORYDEVICE_CUDA);
		Vector3f lightSource = ComputeLightSource(invM);

		dim3 cudaBlockSize(16, 12);
		dim3 gridSize((int)ceil((float)imgSize.x / (float)cudaBlockSize.x), (int)ceil((float)imgSize.y / (float)cudaBlockSize.y));
		renderICP_device<TVoxel, TIndex> <<<gridSize, cudaBlockSize>>>(outRendering, pointsMap, normalsMap, pointsRay,
			scene->localVBA.GetVoxelBlocks(), scene->index.getIndexData(), scene->sceneParams->voxelSize, imgSize, lightSource);
	}

	ITMSafeCall(hipDeviceSynchronize());
}

template<class TVoxel, class TIndex>
void ITMVisualisationEngine_CUDA<TVoxel, TIndex>::RenderImage(const ITMScene<TVoxel, TIndex> *scene, const ITMPose *pose, const ITMIntrinsics *intrinsics,
	const ITMRenderState *renderState, ITMUChar4Image *outputImage, bool useColour)
{
	RenderImage_common(scene, pose, intrinsics, renderState, outputImage, useColour);
}

template<class TVoxel>
void ITMVisualisationEngine_CUDA<TVoxel, ITMVoxelBlockHash>::RenderImage(const ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMPose *pose,
	const ITMIntrinsics *intrinsics, const ITMRenderState *renderState, ITMUChar4Image *outputImage, bool useColour)
{
	RenderImage_common(scene, pose, intrinsics, renderState, outputImage, useColour);
}

template<class TVoxel, class TIndex>
void ITMVisualisationEngine_CUDA<TVoxel, TIndex>::FindSurface(const ITMScene<TVoxel, TIndex> *scene, const ITMPose *pose, const ITMIntrinsics *intrinsics,
	const ITMRenderState *renderState)
{
	FindSurface_common(scene, pose, intrinsics, renderState);
}

template<class TVoxel>
void ITMVisualisationEngine_CUDA<TVoxel, ITMVoxelBlockHash>::FindSurface(const ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMPose *pose,
	const ITMIntrinsics *intrinsics, const ITMRenderState *renderState)
{
	FindSurface_common(scene, pose, intrinsics, renderState);
}

template<class TVoxel, class TIndex>
void ITMVisualisationEngine_CUDA<TVoxel, TIndex>::CreatePointCloud(const ITMScene<TVoxel, TIndex> *scene, const ITMView *view, ITMTrackingState *trackingState, 
	ITMRenderState *renderState, bool skipPoints)
{
	CreatePointCloud_common(scene, view, trackingState, renderState, skipPoints, noTotalPoints_device);
}

template<class TVoxel>
void ITMVisualisationEngine_CUDA<TVoxel, ITMVoxelBlockHash>::CreatePointCloud(const ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view,
	ITMTrackingState *trackingState, ITMRenderState *renderState, bool skipPoints)
{
	CreatePointCloud_common(scene, view, trackingState, renderState, skipPoints, noTotalPoints_device);
}

template<class TVoxel, class TIndex>
void ITMVisualisationEngine_CUDA<TVoxel, TIndex>::CreateICPMaps(const ITMScene<TVoxel, TIndex> *scene, const ITMView *view, ITMTrackingState *trackingState,
	ITMRenderState *renderState)
{
	CreateICPMaps_common(scene, view, trackingState, renderState);
}

template<class TVoxel>
void ITMVisualisationEngine_CUDA<TVoxel, ITMVoxelBlockHash>::CreateICPMaps(const ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view,
	ITMTrackingState *trackingState, ITMRenderState *renderState)
{
	CreateICPMaps_common(scene, view, trackingState, renderState);
}

//device implementations

__global__ void buildVisibleList_device(const ITMHashEntry *hashTable, /*ITMHashCacheState *cacheStates, bool useSwapping,*/ int noTotalEntries,
	int *visibleEntryIDs, int *noVisibleEntries, uchar *entriesVisibleType, Matrix4f M, Vector4f projParams, Vector2i imgSize, float voxelSize)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	__shared__ bool shouldPrefix;

	unsigned char hashVisibleType = 0; //entriesVisibleType[targetIdx];
	const ITMHashEntry &hashEntry = hashTable[targetIdx];

	shouldPrefix = false;
	__syncthreads();

	if (hashEntry.ptr >= 0)
	{
		shouldPrefix = true;

		bool isVisible, isVisibleEnlarged;
		checkBlockVisibility<false>(isVisible, isVisibleEnlarged, hashEntry.pos, M, projParams, voxelSize, imgSize);

		hashVisibleType = isVisible;
	}

	if (hashVisibleType > 0) shouldPrefix = true;

	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType > 0, noVisibleEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) visibleEntryIDs[offset] = targetIdx;
	}
}

template<typename T> __global__ void memsetKernel_device(T *devPtr, const T val, size_t nwords)
{
	size_t offset = threadIdx.x + blockDim.x * blockIdx.x;
	if (offset >= nwords) return;
	devPtr[offset] = val;
}

__global__ void projectAndSplitBlocks_device(const ITMHashEntry *hashEntries, const int *visibleEntryIDs, int noVisibleEntries,
	const Matrix4f pose_M, const Vector4f intrinsics, const Vector2i imgSize, float voxelSize, RenderingBlock *renderingBlocks,
	uint *noTotalBlocks)
{
	int in_offset = threadIdx.x + blockDim.x * blockIdx.x;

	const ITMHashEntry & blockData(hashEntries[visibleEntryIDs[in_offset]]);

	Vector2i upperLeft, lowerRight;
	Vector2f zRange;
	bool validProjection = false;
	if (in_offset < noVisibleEntries) if (blockData.ptr >= 0)
		validProjection = ProjectSingleBlock(blockData.pos, pose_M, intrinsics, imgSize, voxelSize, upperLeft, lowerRight, zRange);

	Vector2i requiredRenderingBlocks(ceilf((float)(lowerRight.x - upperLeft.x + 1) / renderingBlockSizeX),
		ceilf((float)(lowerRight.y - upperLeft.y + 1) / renderingBlockSizeY));

	size_t requiredNumBlocks = requiredRenderingBlocks.x * requiredRenderingBlocks.y;
	if (!validProjection) requiredNumBlocks = 0;

	int out_offset = computePrefixSum_device<uint>(requiredNumBlocks, noTotalBlocks, blockDim.x, threadIdx.x);
	if (!validProjection) return;
	if ((out_offset == -1) || (out_offset + requiredNumBlocks > MAX_RENDERING_BLOCKS)) return;

	CreateRenderingBlocks(renderingBlocks, out_offset, upperLeft, lowerRight, zRange);
}

__global__ void fillBlocks_device(const uint *noTotalBlocks, const RenderingBlock *renderingBlocks,
	Vector2i imgSize, Vector2f *minmaxData)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	int block = blockIdx.x * 4 + blockIdx.y;
	if (block >= *noTotalBlocks) return;

	const RenderingBlock & b(renderingBlocks[block]);
	int xpos = b.upperLeft.x + x;
	if (xpos > b.lowerRight.x) return;
	int ypos = b.upperLeft.y + y;
	if (ypos > b.lowerRight.y) return;

	Vector2f & pixel(minmaxData[xpos + ypos*imgSize.x]);
	atomicMin(&pixel.x, b.zRange.x); atomicMax(&pixel.y, b.zRange.y);
}

template<class TVoxel, class TIndex>
__global__ void genericRaycast_device(Vector4f *out_ptsRay, const TVoxel *voxelData, const typename TIndex::IndexData *voxelIndex,
	Vector2i imgSize, Matrix4f invM, Vector4f projParams, float oneOverVoxelSize, const Vector2f *minmaxdata, float mu)
{
	int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

	if (x >= imgSize.x || y >= imgSize.y) return;

	int locId = x + y * imgSize.x;
	int locId2 = (int)floor((float)x / minmaximg_subsample) + (int)floor((float)y / minmaximg_subsample) * imgSize.x;

	castRay<TVoxel, TIndex>(out_ptsRay[locId], x, y, voxelData, voxelIndex, invM, projParams, oneOverVoxelSize, mu, minmaxdata[locId2]);
}

template<class TVoxel, class TIndex>
__global__ void renderICP_device(Vector4u *outRendering, Vector4f *pointsMap, Vector4f *normalsMap, const Vector4f *ptsRay,
	const TVoxel *voxelData, const typename TIndex::IndexData *voxelIndex, float voxelSize, Vector2i imgSize, Vector3f lightSource)
{
	int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

	if (x >= imgSize.x || y >= imgSize.y) return;

	int locId = x + y * imgSize.x;

	Vector4f ptRay = ptsRay[locId];

	processPixelICP<TVoxel, TIndex>(outRendering[locId], pointsMap[locId], normalsMap[locId], ptRay.toVector3(), ptRay.w > 0, voxelData,
		voxelIndex, voxelSize, lightSource);
}

template<class TVoxel, class TIndex>
__global__ void renderGrey_device(Vector4u *outRendering, const Vector4f *ptsRay, const TVoxel *voxelData,
	const typename TIndex::IndexData *voxelIndex, Vector2i imgSize, Vector3f lightSource)
{
	int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

	if (x >= imgSize.x || y >= imgSize.y) return;

	int locId = x + y * imgSize.x;

	Vector4f ptRay = ptsRay[locId];

	processPixelGrey<TVoxel, TIndex>(outRendering[locId], ptRay.toVector3(), ptRay.w > 0, voxelData, voxelIndex, lightSource);
}

template<class TVoxel, class TIndex>
__global__ void renderPointCloud_device(Vector4u *outRendering, Vector4f *locations, Vector4f *colours, uint *noTotalPoints,
	const Vector4f *ptsRay, const TVoxel *voxelData, const typename TIndex::IndexData *voxelIndex, bool skipPoints,
	float voxelSize, Vector2i imgSize, Vector3f lightSource)
{
	__shared__ bool shouldPrefix;
	shouldPrefix = false;
	__syncthreads();

	bool foundPoint = false; Vector3f point(0.0f);

	int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

	if (x < imgSize.x && y < imgSize.y)
	{
		int locId = x + y * imgSize.x;
		Vector3f outNormal; float angle; Vector4f pointRay;

		pointRay = ptsRay[locId];
		point = pointRay.toVector3();
		foundPoint = pointRay.w > 0;

		computeNormalAndAngle<TVoxel, TIndex>(foundPoint, point, voxelData, voxelIndex, lightSource, outNormal, angle);

		if (foundPoint) drawPixelGrey(outRendering[locId], angle);
		else outRendering[locId] = Vector4u((uchar)0);

		if (skipPoints && ((x % 2 == 0) || (y % 2 == 0))) foundPoint = false;

		if (foundPoint) shouldPrefix = true;
	}

	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<uint>(foundPoint, noTotalPoints, blockDim.x * blockDim.y, threadIdx.x + threadIdx.y * blockDim.x);

		if (offset != -1)
		{
			Vector4f tmp;
			tmp = VoxelColorReader<TVoxel::hasColorInformation, TVoxel, TIndex>::interpolate(voxelData, voxelIndex, point);
			if (tmp.w > 0.0f) { tmp.x /= tmp.w; tmp.y /= tmp.w; tmp.z /= tmp.w; tmp.w = 1.0f; }
			colours[offset] = tmp;

			Vector4f pt_ray_out;
			pt_ray_out.x = point.x * voxelSize; pt_ray_out.y = point.y * voxelSize;
			pt_ray_out.z = point.z * voxelSize; pt_ray_out.w = 1.0f;
			locations[offset] = pt_ray_out;
		}
	}
}

template<class TVoxel, class TIndex>
__global__ void renderColour_device(Vector4u *outRendering, const Vector4f *ptsRay, const TVoxel *voxelData,
	const typename TIndex::IndexData *voxelIndex, Vector2i imgSize, Vector3f lightSource)
{
	int x = (threadIdx.x + blockIdx.x * blockDim.x), y = (threadIdx.y + blockIdx.y * blockDim.y);

	if (x >= imgSize.x || y >= imgSize.y) return;

	int locId = x + y * imgSize.x;

	Vector4f ptRay = ptsRay[locId];

	processPixelColour<TVoxel, TIndex>(outRendering[locId], ptRay.toVector3(), ptRay.w > 0, voxelData, voxelIndex, lightSource);
}

template class ITMLib::Engine::ITMVisualisationEngine_CUDA < ITMVoxel, ITMVoxelIndex > ;
