#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSurfelVisualisationEngine_CUDA.h"

#include "../../DeviceAgnostic/ITMSurfelVisualisationEngine.h"

namespace ITMLib
{

//#################### CUDA KERNELS ####################

__global__ void ck_clear_surfel_index_image(int pixelCount, unsigned int *surfelIndexImage, int *depthBuffer)
{
  int locId = threadIdx.x + blockDim.x * blockIdx.x;
  if(locId < pixelCount)
  {
    clear_surfel_index_image(locId, surfelIndexImage, depthBuffer);
  }
}

#if DEBUG_CORRESPONDENCES
template <typename TSurfel>
__global__ void ck_copy_correspondences_to_buffer(int surfelCount, const TSurfel *surfels, float *correspondences)
{
  int surfelId = threadIdx.x + blockDim.x * blockIdx.x;
  if(surfelId < surfelCount)
  {
    copy_correspondences_to_buffer(surfelId, surfels, correspondences);
  }
}
#endif

template <typename TSurfel>
__global__ void ck_copy_scene_to_buffers(int surfelCount, const TSurfel *surfels, float *positions, unsigned char *normals, unsigned char *colours)
{
  int surfelId = threadIdx.x + blockDim.x * blockIdx.x;
  if(surfelId < surfelCount)
  {
    copy_surfel_to_buffers(surfelId, surfels, positions, normals, colours);
  }
}

template <typename TSurfel>
__global__ void ck_project_to_surfel_index_image(int surfelCount, const TSurfel *surfels, Matrix4f invT, ITMIntrinsics intrinsics, int indexImageWidth, int indexImageHeight,
                                                 int scaleFactor, unsigned int *surfelIndexImage, int *depthBuffer)
{
  int surfelId = threadIdx.x + blockDim.x * blockIdx.x;
  if(surfelId < surfelCount)
  {
    project_to_surfel_index_image(surfelId, surfels, invT, intrinsics, indexImageWidth, indexImageHeight, scaleFactor, surfelIndexImage, depthBuffer);
  }
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

#if DEBUG_CORRESPONDENCES
template <typename TSurfel>
void ITMSurfelVisualisationEngine_CUDA<TSurfel>::CopyCorrespondencesToBuffer(const ITMSurfelScene<TSurfel> *scene, float *correspondences) const
{
  const int surfelCount = static_cast<int>(scene->GetSurfelCount());

  int threadsPerBlock = 256;
  int numBlocks = (surfelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_copy_correspondences_to_buffer<<<numBlocks,threadsPerBlock>>>(
    surfelCount,
    scene->GetSurfels()->GetData(MEMORYDEVICE_CUDA),
    correspondences
  );
}
#endif

template <typename TSurfel>
void ITMSurfelVisualisationEngine_CUDA<TSurfel>::CopySceneToBuffers(const ITMSurfelScene<TSurfel> *scene, float *positions, unsigned char *normals, unsigned char *colours) const
{
  const int surfelCount = static_cast<int>(scene->GetSurfelCount());

  int threadsPerBlock = 256;
  int numBlocks = (surfelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_copy_scene_to_buffers<<<numBlocks,threadsPerBlock>>>(
    surfelCount,
    scene->GetSurfels()->GetData(MEMORYDEVICE_CUDA),
    positions,
    normals,
    colours
  );
}

template <typename TSurfel>
void ITMSurfelVisualisationEngine_CUDA<TSurfel>::RenderImage(const ITMSurfelScene<TSurfel> *scene, const ITMPose *pose, const ITMIntrinsics *intrinsics,
                                                             const ITMSurfelRenderState *renderState, ITMUChar4Image *outputImage, RenderImageType type) const
{
  // TODO
}

//#################### PRIVATE MEMBER FUNCTIONS ####################

template <typename TSurfel>
MemoryDeviceType ITMSurfelVisualisationEngine_CUDA<TSurfel>::GetMemoryType() const
{
  return MEMORYDEVICE_CUDA;
}

template <typename TSurfel>
void ITMSurfelVisualisationEngine_CUDA<TSurfel>::MakeIndexImage(const ITMSurfelScene<TSurfel> *scene, const ITMPose *pose, const ITMIntrinsics *intrinsics,
                                                                int width, int height, int scaleFactor, unsigned int *surfelIndexImage, int *depthBuffer) const
{
  const int pixelCount = width * height;

  int threadsPerBlock = 256;
  int numBlocks = (pixelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_clear_surfel_index_image<<<numBlocks,threadsPerBlock>>>(
    pixelCount,
    surfelIndexImage,
    depthBuffer
  );

  const int surfelCount = static_cast<int>(scene->GetSurfelCount());
  numBlocks = (surfelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_project_to_surfel_index_image<<<numBlocks,threadsPerBlock>>>(
    surfelCount,
    scene->GetSurfels()->GetData(MEMORYDEVICE_CUDA),
    pose->GetM(),
    *intrinsics,
    width,
    height,
    scaleFactor,
    surfelIndexImage,
    depthBuffer
  );
}

//#################### EXPLICIT INSTANTIATIONS ####################

template class ITMSurfelVisualisationEngine_CUDA<ITMSurfel>;
template class ITMSurfelVisualisationEngine_CUDA<ITMSurfel_rgb>;

}
