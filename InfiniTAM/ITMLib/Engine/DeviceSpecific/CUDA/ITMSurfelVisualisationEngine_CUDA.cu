#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSurfelVisualisationEngine_CUDA.h"

#include "../../DeviceAgnostic/ITMSurfelVisualisationEngine.h"

namespace ITMLib
{

//#################### CUDA KERNELS ####################

template <typename TSurfel>
__global__ void ck_copy_scene_to_buffers(int surfelCount, const TSurfel *surfels, float *positions)
{
  int surfelId = threadIdx.x + blockDim.x * blockIdx.x;
  if(surfelId < surfelCount)
  {
    copy_surfel_to_buffers(surfelId, surfels, positions);
  }
}

//#################### PUBLIC MEMBER FUNCTIONS ####################

template <typename TSurfel>
void ITMSurfelVisualisationEngine_CUDA<TSurfel>::CopySceneToBuffers(const ITMSurfelScene<TSurfel> *scene, float *positions) const
{
  const int surfelCount = static_cast<int>(scene->GetSurfelCount());

  int threadsPerBlock = 256;
  int numBlocks = (surfelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_copy_scene_to_buffers<<<numBlocks,threadsPerBlock>>>(
    surfelCount,
    scene->GetSurfels()->GetData(MEMORYDEVICE_CUDA),
    positions
  );
}

template <typename TSurfel>
void ITMSurfelVisualisationEngine_CUDA<TSurfel>::FindSurface(const ITMSurfelScene<TSurfel> *scene, const ITMPose *pose, const ITMIntrinsics *intrinsics,
                                                             const ITMSurfelRenderState *renderState) const
{
  // TODO
}

template <typename TSurfel>
void ITMSurfelVisualisationEngine_CUDA<TSurfel>::RenderImage(const ITMSurfelScene<TSurfel> *scene, const ITMPose *pose, const ITMIntrinsics *intrinsics,
                                                             const ITMSurfelRenderState *renderState, ITMUChar4Image *outputImage, RenderImageType type) const
{
  // TODO
}

//#################### EXPLICIT INSTANTIATIONS ####################

template class ITMSurfelVisualisationEngine_CUDA<ITMSurfel>;

}
