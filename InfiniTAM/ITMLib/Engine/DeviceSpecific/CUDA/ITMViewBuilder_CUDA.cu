#include "hip/hip_runtime.h"
// Copyright 2014 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMViewBuilder_CUDA.h"
#include "../../../../ORUtils/CUDADefines.h"

#include "../../DeviceAgnostic/ITMViewBuilder.h"
#include "../../../../ORUtils/MemoryBlock.h"

using namespace ITMLib::Engine;
using namespace ORUtils;

ITMViewBuilder_CUDA::ITMViewBuilder_CUDA(const ITMRGBDCalib *calib):ITMViewBuilder(calib) { }
ITMViewBuilder_CUDA::~ITMViewBuilder_CUDA(void) { }

__global__ void convertDisparityToDepth_device(float *depth_out, const short *depth_in, Vector2f disparityCalibParams, float fx_depth, Vector2i imgSize);
__global__ void convertDepthMMToFloat_device(float *d_out, const short *d_in, Vector2i imgSize);

// host methods

void ITMViewBuilder_CUDA::UpdateView(ITMView **view_ptr, ITMUChar4Image *rgbImage, ITMShortImage *rawDepthImage)
{
	if (*view_ptr == NULL)
	{
		*view_ptr = new ITMView(calib, rgbImage->noDims, rawDepthImage->noDims, true);
		if (this->shortImage != NULL) delete this->shortImage;
		this->shortImage = new ITMShortImage(rawDepthImage->noDims, true, true);
	}

	ITMView *view = *view_ptr;

	view->rgb->SetFrom(rgbImage, MemoryBlock<Vector4u>::CPU_TO_CUDA);
	this->shortImage->SetFrom(rawDepthImage, MemoryBlock<short>::CPU_TO_CUDA);

	switch (inputImageType)
	{
	case InfiniTAM_DISPARITY_IMAGE:
		this->ConvertDisparityToDepth(view->depth, this->shortImage, &(view->calib->intrinsics_d), &(view->calib->disparityCalib));
		break;
	case InfiniTAM_SHORT_DEPTH_IMAGE:
		this->ConvertDepthMMToFloat(view->depth, this->shortImage);
		break;
	default:
		break;
	}
}

void ITMViewBuilder_CUDA::UpdateView(ITMView **view_ptr, ITMUChar4Image *rgbImage, ITMFloatImage *depthImage)
{
	if (*view_ptr == NULL)
		*view_ptr = new ITMView(calib, rgbImage->noDims, depthImage->noDims, true);

	ITMView *view = *view_ptr;

	view->rgb->UpdateDeviceFromHost();
	view->depth->UpdateDeviceFromHost();
}

void ITMViewBuilder_CUDA::UpdateView(ITMView **view_ptr, ITMUChar4Image *rgbImage, ITMShortImage *depthImage, ITMIMUMeasurement *imuMeasurement)
{
	if (*view_ptr == NULL) 
	{
		*view_ptr = new ITMViewIMU(calib, rgbImage->noDims, depthImage->noDims, true);
		if (this->shortImage != NULL) delete this->shortImage;
		this->shortImage = new ITMShortImage(depthImage->noDims, true, true);
	}

	ITMViewIMU* imuView = (ITMViewIMU*)(*view_ptr);
	imuView->imu->SetFrom(imuMeasurement);

	this->UpdateView(view_ptr, rgbImage, depthImage);
}

void ITMViewBuilder_CUDA::ConvertDisparityToDepth(ITMFloatImage *depth_out, const ITMShortImage *depth_in, const ITMIntrinsics *depthIntrinsics,
	const ITMDisparityCalib *disparityCalib)
{
	Vector2i imgSize = depth_in->noDims;

	const short *d_in = depth_in->GetData(MEMORYDEVICE_CUDA);
	float *d_out = depth_out->GetData(MEMORYDEVICE_CUDA);

	Vector2f disparityCalibParams; float fx_depth;
	disparityCalibParams.x = disparityCalib->params.x;
	disparityCalibParams.y = disparityCalib->params.y;
	fx_depth = depthIntrinsics->projectionParamsSimple.fx;

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	convertDisparityToDepth_device << <gridSize, blockSize >> >(d_out, d_in, disparityCalibParams, fx_depth, imgSize);
}

void ITMViewBuilder_CUDA::ConvertDepthMMToFloat(ITMFloatImage *depth_out, const ITMShortImage *depth_in)
{
	Vector2i imgSize = depth_in->noDims;

	const short *d_in = depth_in->GetData(MEMORYDEVICE_CUDA);
	float *d_out = depth_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	convertDepthMMToFloat_device << <gridSize, blockSize >> >(d_out, d_in, imgSize);
}

// device functions

__global__ void convertDisparityToDepth_device(float *d_out, const short *d_in, Vector2f disparityCalibParams, float fx_depth, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= imgSize.x) || (y >= imgSize.y)) return;

	convertDisparityToDepth(d_out, x, y, d_in, disparityCalibParams, fx_depth, imgSize);
}

__global__ void convertDepthMMToFloat_device(float *d_out, const short *d_in, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= imgSize.x) || (y >= imgSize.y)) return;

	convertDepthMMToFloat(d_out, x, y, d_in, imgSize);
}
