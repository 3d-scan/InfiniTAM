#include "hip/hip_runtime.h"
// Copyright 2014 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMViewBuilder_CUDA.h"
#include "../../../../ORUtils/CUDADefines.h"

#include "../../DeviceAgnostic/ITMViewBuilder.h"

using namespace ITMLib::Engine;

ITMViewBuilder_CUDA::ITMViewBuilder_CUDA(const ITMRGBDCalib *calib, ITMLibSettings::DeviceType deviceType)
	:ITMViewBuilder(calib, deviceType) { }
ITMViewBuilder_CUDA::~ITMViewBuilder_CUDA(void) { }

__global__ void convertDisparityToDepth_device(float *depth_out, const short *depth_in, Vector2f disparityCalibParams, float fx_depth, Vector2i imgSize);
__global__ void convertDepthMMToFloat_device(float *d_out, const short *d_in, Vector2i imgSize);

// host methods

void ITMViewBuilder_CUDA::ConvertDisparityToDepth(ITMFloatImage *depth_out, const ITMShortImage *depth_in, const ITMIntrinsics *depthIntrinsics,
	const ITMDisparityCalib *disparityCalib)
{
	Vector2i imgSize = depth_in->noDims;

	const short *d_in = depth_in->GetData(MEMORYDEVICE_CUDA);
	float *d_out = depth_out->GetData(MEMORYDEVICE_CUDA);

	Vector2f disparityCalibParams; float fx_depth;
	disparityCalibParams.x = disparityCalib->params.x;
	disparityCalibParams.y = disparityCalib->params.y;
	fx_depth = depthIntrinsics->projectionParamsSimple.fx;

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	convertDisparityToDepth_device << <gridSize, blockSize >> >(d_out, d_in, disparityCalibParams, fx_depth, imgSize);
}

void ITMViewBuilder_CUDA::ConvertDepthMMToFloat(ITMFloatImage *depth_out, const ITMShortImage *depth_in)
{
	Vector2i imgSize = depth_in->noDims;

	const short *d_in = depth_in->GetData(MEMORYDEVICE_CUDA);
	float *d_out = depth_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	convertDepthMMToFloat_device << <gridSize, blockSize >> >(d_out, d_in, imgSize);
}

// device functions

__global__ void convertDisparityToDepth_device(float *d_out, const short *d_in, Vector2f disparityCalibParams, float fx_depth, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= imgSize.x) || (y >= imgSize.y)) return;

	convertDisparityToDepth(d_out, x, y, d_in, disparityCalibParams, fx_depth, imgSize);
}

__global__ void convertDepthMMToFloat_device(float *d_out, const short *d_in, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= imgSize.x) || (y >= imgSize.y)) return;

	convertDepthMMToFloat(d_out, x, y, d_in, imgSize);
}