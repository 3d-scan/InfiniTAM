#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSurfelSceneReconstructionEngine_CUDA.h"

#include "../../DeviceAgnostic/ITMSurfelSceneReconstructionEngine.h"

#define DEBUGGING 1

namespace ITMLib
{

//#################### CUDA KERNELS ####################

template <typename TSurfel>
__global__ void ck_project_to_index_map(int surfelCount, const TSurfel *surfels, Matrix4f invT, ITMIntrinsics intrinsics, int depthMapWidth, int depthMapHeight,
                                        unsigned int *indexMap)
{
  int surfelId = threadIdx.x + blockDim.x * blockIdx.x;
  if(surfelId < surfelCount)
  {
    project_to_index_map(surfelId, surfels, invT, intrinsics, depthMapWidth, depthMapHeight, indexMap);
  }
}

__global__ void ck_calculate_vertex_position(int pixelCount, int width, ITMIntrinsics intrinsics, const float *depthMap, Vector3f *vertexMap)
{
  int locId = threadIdx.x + blockDim.x * blockIdx.x;
  if(locId < pixelCount)
  {
    calculate_vertex_position(locId, width, intrinsics, depthMap, vertexMap);
  }
}

//#################### CONSTRUCTORS ####################

template <typename TSurfel>
ITMSurfelSceneReconstructionEngine_CUDA<TSurfel>::ITMSurfelSceneReconstructionEngine_CUDA(const Vector2i& depthImageSize)
: ITMSurfelSceneReconstructionEngine<TSurfel>(depthImageSize)
{}

//#################### PUBLIC MEMBER FUNCTIONS ####################

template <typename TSurfel>
void ITMSurfelSceneReconstructionEngine_CUDA<TSurfel>::AllocateSceneFromDepth(ITMSurfelScene<TSurfel> *scene, const ITMView *view, const ITMTrackingState *trackingState) const
{
  // TODO
}

template <typename TSurfel>
void ITMSurfelSceneReconstructionEngine_CUDA<TSurfel>::IntegrateIntoScene(ITMSurfelScene<TSurfel> *scene, const ITMView *view, const ITMTrackingState *trackingState) const
{
  // TEMPORARY
  PreprocessDepthMap(view);
  //GenerateIndexMap(scene, view, *trackingState->pose_d);

  // TODO
}

//#################### PRIVATE MEMBER FUNCTIONS ####################

template <typename TSurfel>
void ITMSurfelSceneReconstructionEngine_CUDA<TSurfel>::GenerateIndexMap(const ITMSurfelScene<TSurfel> *scene, const ITMView *view, const ITMPose& pose) const
{
  const int surfelCount = static_cast<int>(scene->GetSurfelCount());

  int threadsPerBlock = 256;
  int numBlocks = (surfelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_project_to_index_map<<<numBlocks,threadsPerBlock>>>(
    surfelCount,
    scene->GetSurfels()->GetData(MEMORYDEVICE_CUDA),
    pose.GetInvM(),
    view->calib->intrinsics_d,
    view->depth->noDims.x,
    view->depth->noDims.y,
    m_indexMap->GetData(MEMORYDEVICE_CUDA)
  );

#if DEBUGGING
  m_indexMap->UpdateHostFromDevice();
#endif
}

template <typename TSurfel>
void ITMSurfelSceneReconstructionEngine_CUDA<TSurfel>::PreprocessDepthMap(const ITMView *view) const
{
  const int pixelCount = static_cast<int>(view->depth->dataSize);

  // Calculate the vertex map.
  int threadsPerBlock = 256;
  int numBlocks = (pixelCount + threadsPerBlock - 1) / threadsPerBlock;

  ck_calculate_vertex_position<<<numBlocks,threadsPerBlock>>>(
    pixelCount,
    view->depth->noDims.x,
    view->calib->intrinsics_d,
    view->depth->GetData(MEMORYDEVICE_CUDA),
    m_vertexMap->GetData(MEMORYDEVICE_CUDA)
  );

#if DEBUGGING
  m_vertexMap->UpdateHostFromDevice();
#endif

  // Calculate the normal map.
  // FIXME: We don't need to store two copies of it.
  m_normalMap->SetFrom(view->depthNormal, ORUtils::MemoryBlock<Vector4f>::CUDA_TO_CUDA);

#if DEBUGGING
  m_normalMap->UpdateHostFromDevice();
#endif

  // TODO: Calculate the radius map.
}

//#################### EXPLICIT INSTANTIATIONS ####################

template class ITMSurfelSceneReconstructionEngine_CUDA<ITMSurfel>;

}
