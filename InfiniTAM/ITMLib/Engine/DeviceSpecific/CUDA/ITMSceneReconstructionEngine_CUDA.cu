#include "hip/hip_runtime.h"
// Copyright 2014 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMSceneReconstructionEngine_CUDA.h"
#include "ITMCUDAUtils.h"
#include "../../DeviceAgnostic/ITMSceneReconstructionEngine.h"
#include "../../../Objects/ITMRenderState_VH.h"

using namespace ITMLib::Engine;

template<class TVoxel>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *noLiveEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i imgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

template<class TVoxel>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW);

template<class TVoxel>
__global__ void integrateIntoSceneH_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *liveEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb,
	Vector4f projParams_d, Vector4f projParams_rgb, const ITMVoxelBlockHHash::IndexData *indexData,float smallestVoxelSize, float mu, int maxW);

__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
	float viewFrustrum_max);
__global__ void buildHHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHHashEntry *hashTable, float viewFrustum_min,
	float viewFrustrum_max);

__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	int *noAllocatedVoxelEntries, int *noAllocatedExcessEntries, uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords);

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	int *noAllocatedVoxelEntries, uchar *entriesVisibleType);

template<bool useSwapping>
__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashCacheState *cacheStates, int noTotalEntries, 
	int *liveEntryIDs, int *noLiveEntries, uchar *entriesVisibleType, Matrix4f M_d, Vector4f projParams_d, Vector2i imgSize, float voxelSize, int offsetToAdd);

// host methods

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipMalloc((void**)&noLiveEntries_device, sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&noAllocatedVoxelEntries_device, sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&noAllocatedExcessEntries_device, sizeof(int)));

	int noTotalEntries = ITMVoxelBlockHash::noVoxelBlocks;
	ITMSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ITMSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector3s)));
}

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel,ITMVoxelBlockHash>::~ITMSceneReconstructionEngine_CUDA(void) 
{
	ITMSafeCall(hipFree(noLiveEntries_device));
	ITMSafeCall(hipFree(noAllocatedVoxelEntries_device));
	ITMSafeCall(hipFree(noAllocatedExcessEntries_device));

	ITMSafeCall(hipFree(entriesAllocType_device));
	ITMSafeCall(hipFree(blockCoords_device));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view, 
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	M_d = trackingState->pose_d->M; M_d.inv(invM_d);

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	ITMHashEntry *hashTable = scene->index.GetEntries();
	ITMHashCacheState *cacheStates = scene->useSwapping ? scene->globalCache->GetCacheStates(true) : 0;

	int noTotalEntries = scene->index.noVoxelBlocks;
	int lastFreeExcessListId = scene->index.GetLastFreeExcessListId();

	int *liveEntryIDs = renderState_vh->GetLiveEntryIDs();
	uchar *entriesVisibleType = renderState_vh->GetEntriesVisibleType();

	float oneOverVoxelSize = 1.0f / (voxelSize * SDF_BLOCK_SIZE);

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));

	ITMSafeCall(hipMemcpy(noAllocatedVoxelEntries_device, &scene->localVBA.lastFreeBlockId, sizeof(int), hipMemcpyHostToDevice));
	ITMSafeCall(hipMemcpy(noAllocatedExcessEntries_device, &lastFreeExcessListId, sizeof(int), hipMemcpyHostToDevice));
	ITMSafeCall(hipMemset(noLiveEntries_device, 0, sizeof(int)));

	ITMSafeCall(hipMemset(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));
	ITMSafeCall(hipMemset(entriesVisibleType, 0, sizeof(unsigned char)* noTotalEntries));

	buildHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType, 
		blockCoords_device, depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverVoxelSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max);

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));

	allocateVoxelBlocksList_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, excessAllocationList, hashTable,
		noTotalEntries, noAllocatedVoxelEntries_device, noAllocatedExcessEntries_device, entriesAllocType_device, entriesVisibleType, 
		blockCoords_device);

	if (scene->useSwapping)
		buildVisibleList_device<true> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, cacheStates, noTotalEntries, liveEntryIDs,
		noLiveEntries_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize, 0);
	else
		buildVisibleList_device<false> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable, cacheStates, noTotalEntries, liveEntryIDs,
		noLiveEntries_device, entriesVisibleType, M_d, projParams_d, depthImgSize, voxelSize, 0);

	if (scene->useSwapping)
	{
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries, 
			noAllocatedVoxelEntries_device, entriesVisibleType);
	}

	ITMSafeCall(hipMemcpy(&renderState_vh->noLiveEntries, noLiveEntries_device, sizeof(int), hipMemcpyDeviceToHost));
	ITMSafeCall(hipMemcpy(&scene->localVBA.lastFreeBlockId, noAllocatedVoxelEntries_device, sizeof(int), hipMemcpyDeviceToHost));
	ITMSafeCall(hipMemcpy(&lastFreeExcessListId, noAllocatedExcessEntries_device, sizeof(int), hipMemcpyDeviceToHost));

	scene->index.SetLastFreeExcessListId(lastFreeExcessListId);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHash>::IntegrateIntoScene(ITMScene<TVoxel, ITMVoxelBlockHash> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	M_d = trackingState->pose_d->M;
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * trackingState->pose_d->M;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHashEntry *hashTable = scene->index.GetEntries();

	int *liveEntryIDs = renderState_vh->GetLiveEntryIDs();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(renderState_vh->noLiveEntries);

	integrateIntoScene_device << <gridSize, cudaBlockSize >> >(localVBA, hashTable, liveEntryIDs,
		rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
}

// plain voxel array

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMPlainVoxelArray>::IntegrateIntoScene(ITMScene<TVoxel, ITMPlainVoxelArray> *scene, const ITMView *view,
	const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float voxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	M_d = trackingState->pose_d->M;
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * trackingState->pose_d->M;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo = scene->index.getIndexData();

	dim3 cudaBlockSize(8, 8, 8);
	dim3 gridSize(scene->index.getVolumeSize().x / cudaBlockSize.x, scene->index.getVolumeSize().y / cudaBlockSize.y, scene->index.getVolumeSize().z / cudaBlockSize.z);

	integrateIntoScene_device << <gridSize, cudaBlockSize >> >(localVBA, arrayInfo,
		rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, voxelSize, mu, maxW);
}

// hierarchical hash

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::ITMSceneReconstructionEngine_CUDA(void)
{
	int noLevels = ITMHHashTable::noLevels;
	ITMSafeCall(hipMalloc((void**)&noLiveEntries_device, sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&noAllocatedVoxelEntries_device, sizeof(int)));
	ITMSafeCall(hipMalloc((void**)&noAllocatedExcessEntries_device, noLevels * sizeof(int)));

	int noTotalEntries = ITMVoxelBlockHHash::noVoxelBlocks;
	ITMSafeCall(hipMalloc((void**)&entriesAllocType_device, noTotalEntries));
	ITMSafeCall(hipMalloc((void**)&blockCoords_device, noTotalEntries * sizeof(Vector3s)));
}

template<class TVoxel>
ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::~ITMSceneReconstructionEngine_CUDA(void)
{
	ITMSafeCall(hipFree(noLiveEntries_device));
	ITMSafeCall(hipFree(noAllocatedVoxelEntries_device));
	ITMSafeCall(hipFree(noAllocatedExcessEntries_device));

	ITMSafeCall(hipFree(entriesAllocType_device));
	ITMSafeCall(hipFree(blockCoords_device));
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::AllocateSceneFromDepth(ITMScene<TVoxel, ITMVoxelBlockHHash> *scene, const ITMView *view, const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i depthImgSize = view->depth->noDims;
	float smallestVoxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, invM_d;
	Vector4f projParams_d, invProjParams_d;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	M_d = trackingState->pose_d->M; M_d.inv(invM_d);

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	invProjParams_d = projParams_d;
	invProjParams_d.x = 1.0f / invProjParams_d.x;
	invProjParams_d.y = 1.0f / invProjParams_d.y;

	float mu = scene->sceneParams->mu;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	int *voxelAllocationList = scene->localVBA.GetAllocationList();
	int *excessAllocationList = scene->index.GetExcessAllocationList();
	ITMHHashEntry *hashTable = scene->index.GetEntries();
	ITMHashCacheState *cacheStates = scene->useSwapping ? scene->globalCache->GetCacheStates(true) : 0;

	int noTotalEntries = scene->index.noVoxelBlocks;
	int *lastFreeExcessListIds = scene->index.GetLastFreeExcessListIds();

	int *liveEntryIDs = renderState_vh->GetLiveEntryIDs();
	uchar *entriesVisibleType = renderState_vh->GetEntriesVisibleType();

	float oneOverSmallestVoxelSize = 1.0f / smallestVoxelSize;

	dim3 cudaBlockSizeHV(16, 16);
	dim3 gridSizeHV((int)ceil((float)depthImgSize.x / (float)cudaBlockSizeHV.x), (int)ceil((float)depthImgSize.y / (float)cudaBlockSizeHV.y));
	int noLevels = ITMHHashTable::noLevels;
	int noTotalEntriesPerLevel = ITMHHashTable::noTotalEntriesPerLevel;

	ITMSafeCall(hipMemcpy(noAllocatedVoxelEntries_device, &scene->localVBA.lastFreeBlockId, sizeof(int), hipMemcpyHostToDevice));
	ITMSafeCall(hipMemcpy(noAllocatedExcessEntries_device, lastFreeExcessListIds, noLevels * sizeof(int), hipMemcpyHostToDevice));
	ITMSafeCall(hipMemset(noLiveEntries_device, 0, sizeof(int)));

	ITMSafeCall(hipMemset(entriesAllocType_device, 0, sizeof(unsigned char)* noTotalEntries));
	ITMSafeCall(hipMemset(entriesVisibleType, 0, sizeof(unsigned char)* noTotalEntries));
	ITMSafeCall(hipMemset(blockCoords_device, 0, sizeof(Vector3s)* noTotalEntries));

	buildHHashAllocAndVisibleType_device << <gridSizeHV, cudaBlockSizeHV >> >(entriesAllocType_device, entriesVisibleType,
		blockCoords_device, depth, invM_d, invProjParams_d, mu, depthImgSize, oneOverSmallestVoxelSize, hashTable,
		scene->sceneParams->viewFrustum_min, scene->sceneParams->viewFrustum_max);

	dim3 cudaBlockSizeAL(256, 1);
	dim3 gridSizeAL((int)ceil((float)noTotalEntriesPerLevel / (float)cudaBlockSizeAL.x));

	for (int level = 0; level < noLevels; ++level) {
		int levelOffset = level * noTotalEntriesPerLevel;
		int levelOffsetExcList = level * SDF_EXCESS_LIST_SIZE;

		allocateVoxelBlocksList_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, excessAllocationList + levelOffsetExcList, hashTable + levelOffset,
			noTotalEntriesPerLevel, noAllocatedVoxelEntries_device, noAllocatedExcessEntries_device + level, entriesAllocType_device + levelOffset,
			entriesVisibleType + levelOffset, blockCoords_device + levelOffset);
	}

	cudaBlockSizeAL = dim3(256, 1);
	gridSizeAL = dim3((int)ceil((float)noTotalEntriesPerLevel / (float)cudaBlockSizeAL.x));

	for (int level = 0; level < noLevels; ++level) {
		float voxelSize = smallestVoxelSize * (1 << level);
		int levelOffset = level * noTotalEntriesPerLevel;

		if (scene->useSwapping)
			buildVisibleList_device<true> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable + levelOffset, cacheStates + levelOffset, noTotalEntriesPerLevel, liveEntryIDs, noLiveEntries_device, entriesVisibleType + levelOffset, M_d, projParams_d, depthImgSize, voxelSize, levelOffset);
		else
			buildVisibleList_device<false> << <gridSizeAL, cudaBlockSizeAL >> >(hashTable + levelOffset, cacheStates + levelOffset, noTotalEntriesPerLevel, liveEntryIDs, noLiveEntries_device, entriesVisibleType + levelOffset, M_d, projParams_d, depthImgSize, voxelSize, levelOffset);
	}

	if (scene->useSwapping)
	{
		cudaBlockSizeAL = dim3(256, 1);
		gridSizeAL = dim3((int)ceil((float)noTotalEntries / (float)cudaBlockSizeAL.x));
		reAllocateSwappedOutVoxelBlocks_device << <gridSizeAL, cudaBlockSizeAL >> >(voxelAllocationList, hashTable, noTotalEntries,
			noAllocatedVoxelEntries_device, entriesVisibleType);
	}

	ITMSafeCall(hipMemcpy(&renderState_vh->noLiveEntries, noLiveEntries_device, sizeof(int), hipMemcpyDeviceToHost));
	ITMSafeCall(hipMemcpy(&scene->localVBA.lastFreeBlockId, noAllocatedVoxelEntries_device, sizeof(int), hipMemcpyDeviceToHost));
	ITMSafeCall(hipMemcpy(lastFreeExcessListIds, noAllocatedExcessEntries_device, noLevels * sizeof(int), hipMemcpyDeviceToHost));

	scene->index.SetLastFreeExcessListIds(lastFreeExcessListIds);
}

template<class TVoxel>
void ITMSceneReconstructionEngine_CUDA<TVoxel, ITMVoxelBlockHHash>::IntegrateIntoScene(ITMScene<TVoxel, ITMVoxelBlockHHash> *scene, const ITMView *view, const ITMTrackingState *trackingState, const ITMRenderState *renderState)
{
	Vector2i rgbImgSize = view->rgb->noDims;
	Vector2i depthImgSize = view->depth->noDims;
	float smallestVoxelSize = scene->sceneParams->voxelSize;

	Matrix4f M_d, M_rgb;
	Vector4f projParams_d, projParams_rgb;

	ITMRenderState_VH *renderState_vh = (ITMRenderState_VH*)renderState;

	M_d = trackingState->pose_d->M;
	if (TVoxel::hasColorInformation) M_rgb = view->calib->trafo_rgb_to_depth.calib_inv * trackingState->pose_d->M;

	projParams_d = view->calib->intrinsics_d.projectionParamsSimple.all;
	projParams_rgb = view->calib->intrinsics_rgb.projectionParamsSimple.all;

	float mu = scene->sceneParams->mu; int maxW = scene->sceneParams->maxW;

	float *depth = view->depth->GetData(MEMORYDEVICE_CUDA);
	Vector4u *rgb = view->rgb->GetData(MEMORYDEVICE_CUDA);
	TVoxel *localVBA = scene->localVBA.GetVoxelBlocks();
	ITMHHashEntry *hashTable = scene->index.GetEntries();

	int *liveEntryIDs = renderState_vh->GetLiveEntryIDs();

	dim3 cudaBlockSize(SDF_BLOCK_SIZE, SDF_BLOCK_SIZE, SDF_BLOCK_SIZE);
	dim3 gridSize(renderState_vh->noLiveEntries);

	integrateIntoSceneH_device << <gridSize, cudaBlockSize >> >(localVBA, hashTable, liveEntryIDs,
		rgb, rgbImgSize, depth, depthImgSize, M_d, M_rgb, projParams_d, projParams_rgb, 
		scene->index.getIndexData(), smallestVoxelSize, mu, maxW);
}

// device functions

template<class TVoxel>
__global__ void integrateIntoScene_device(TVoxel *voxelArray, const ITMPlainVoxelArray::ITMVoxelArrayInfo *arrayInfo,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int z = blockIdx.z*blockDim.z+threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * arrayInfo->size.x + z * arrayInfo->size.x * arrayInfo->size.y;

	pt_model.x = (float)(x + arrayInfo->offset.x) * _voxelSize;
	pt_model.y = (float)(y + arrayInfo->offset.y) * _voxelSize;
	pt_model.z = (float)(z + arrayInfo->offset.z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(voxelArray[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

template<class TVoxel>
__global__ void integrateIntoScene_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *liveEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, Vector4f projParams_d, 
	Vector4f projParams_rgb, float _voxelSize, float mu, int maxW)
{
	Vector3i globalPos;
	int entryId = liveEntryIDs[blockIdx.x];

	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	if (currentHashEntry.ptr < 0) return;

	globalPos = currentHashEntry.pos.toInt() * SDF_BLOCK_SIZE;

	TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

	int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	pt_model.x = (float)(globalPos.x + x) * _voxelSize;
	pt_model.y = (float)(globalPos.y + y) * _voxelSize;
	pt_model.z = (float)(globalPos.z + z) * _voxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation,TVoxel>::compute(localVoxelBlock[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

template<class TVoxel>
__global__ void integrateIntoSceneH_device(TVoxel *localVBA, const ITMHashEntry *hashTable, int *liveEntryIDs,
	const Vector4u *rgb, Vector2i rgbImgSize, const float *depth, Vector2i depthImgSize, Matrix4f M_d, Matrix4f M_rgb, 
	Vector4f projParams_d, Vector4f projParams_rgb, const ITMVoxelBlockHHash::IndexData *indexData, float smallestVoxelSize, float mu, int maxW)
{
	Vector3i globalPos;
	int entryId = liveEntryIDs[blockIdx.x];
	const ITMHashEntry &currentHashEntry = hashTable[entryId];

	if (currentHashEntry.ptr < 0) return;

	float localVoxelSize = smallestVoxelSize * (1 << ITMHHashTable::GetLevelForEntry(entryId));
	globalPos.x = currentHashEntry.pos.x;
	globalPos.y = currentHashEntry.pos.y;
	globalPos.z = currentHashEntry.pos.z;
	globalPos *= SDF_BLOCK_SIZE;

	TVoxel *localVoxelBlock = &(localVBA[currentHashEntry.ptr * SDF_BLOCK_SIZE3]);

	int x = threadIdx.x, y = threadIdx.y, z = threadIdx.z;

	Vector4f pt_model; int locId;

	locId = x + y * SDF_BLOCK_SIZE + z * SDF_BLOCK_SIZE * SDF_BLOCK_SIZE;

	pt_model.x = (float)(globalPos.x + x) * localVoxelSize;
	pt_model.y = (float)(globalPos.y + y) * localVoxelSize;
	pt_model.z = (float)(globalPos.z + z) * localVoxelSize;
	pt_model.w = 1.0f;

	ComputeUpdatedVoxelInfo<TVoxel::hasColorInformation, TVoxel>::compute(localVoxelBlock[locId], pt_model, M_d, projParams_d, M_rgb, projParams_rgb, mu, maxW, depth, depthImgSize, rgb, rgbImgSize);
}

__global__ void buildHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHashEntry *hashTable, float viewFrustum_min,
	float viewFrustum_max)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

	buildHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, depth, invM_d,
		projParams_d, mu, _imgSize, _voxelSize, hashTable, viewFrustum_min, viewFrustum_max);
}

__global__ void buildHHashAllocAndVisibleType_device(uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords, const float *depth,
	Matrix4f invM_d, Vector4f projParams_d, float mu, Vector2i _imgSize, float _voxelSize, ITMHHashEntry *hashTable, float viewFrustum_min,
	float viewFrustum_max)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > _imgSize.x - 1 || y > _imgSize.y - 1) return;

	buildHHashAllocAndVisibleTypePP(entriesAllocType, entriesVisibleType, x, y, blockCoords, depth, invM_d,
		projParams_d, mu, _imgSize, _voxelSize, hashTable, viewFrustum_min, viewFrustum_max);
}

__global__ void allocateVoxelBlocksList_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	int *noAllocatedVoxelEntries, int *noAllocatedExcessEntries, uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx, exlIdx;
	ITMHashEntry hashEntry = hashTable[targetIdx];

	switch (entriesAllocType[targetIdx])
	{
	case 1: //needs allocation, fits in the ordered list
		vbaIdx = atomicSub(&noAllocatedVoxelEntries[0], 1);

		if (vbaIdx >= 0) //there is room in the voxel block array
		{
			Vector3s pt_block_all = blockCoords[targetIdx];

			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];

			hashTable[targetIdx] = hashEntry;
		}
		break;

	case 2: //needs allocation in the excess list
		vbaIdx = atomicSub(&noAllocatedVoxelEntries[0], 1);
		exlIdx = atomicSub(&noAllocatedExcessEntries[0], 1);

		if (vbaIdx >= 0 && exlIdx >= 0) //there is room in the voxel block array and excess list
		{
			Vector3s pt_block_all = blockCoords[targetIdx];

			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];

			int exlOffset = excessAllocationList[exlIdx];

			hashTable[targetIdx].offset = exlOffset + 1; //connect to child

			hashTable[SDF_BUCKET_NUM * SDF_ENTRY_NUM_PER_BUCKET + exlOffset] = hashEntry; //add child to the excess list

			entriesVisibleType[SDF_BUCKET_NUM * SDF_ENTRY_NUM_PER_BUCKET + exlOffset] = 1; //make child visible

		}

		break;
	}
}

__global__ void allocateVoxelBlocksListHHash_device(int *voxelAllocationList, int *excessAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	int *noAllocatedVoxelEntries, int *noAllocatedExcessEntries, uchar *entriesAllocType, uchar *entriesVisibleType, Vector3s *blockCoords)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx, exlIdx;
	ITMHashEntry hashEntry = hashTable[targetIdx];

	switch (entriesAllocType[targetIdx])
	{
	case 1: //needs allocation, fits in the ordered list
		vbaIdx = atomicSub(&noAllocatedVoxelEntries[0], 1);

		if (vbaIdx >= 0) //there is room in the voxel block array
		{
			Vector3s pt_block_all = blockCoords[targetIdx];

			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];

			hashTable[targetIdx] = hashEntry;
		}
		break;

	case 2: //needs allocation in the excess list
		int level = ITMHHashTable::GetLevelForEntry(targetIdx);

		vbaIdx = atomicSub(&noAllocatedVoxelEntries[0], 1);
		exlIdx = atomicSub(&noAllocatedExcessEntries[level], 1);

		if (vbaIdx >= 0 && exlIdx >= 0) //there is room in the voxel block array and excess list
		{
			Vector3s pt_block_all = blockCoords[targetIdx];

			hashEntry.pos.x = pt_block_all.x; hashEntry.pos.y = pt_block_all.y; hashEntry.pos.z = pt_block_all.z;
			hashEntry.ptr = voxelAllocationList[vbaIdx];

			int exlOffset = excessAllocationList[level*SDF_EXCESS_LIST_SIZE + exlIdx];

			hashTable[targetIdx].offset = exlOffset + 1; //connect to child

			hashTable[level * ITMHHashTable::noTotalEntriesPerLevel + SDF_BUCKET_NUM * SDF_ENTRY_NUM_PER_BUCKET + exlOffset] = hashEntry; //add child to the excess list

			entriesVisibleType[level * ITMHHashTable::noTotalEntriesPerLevel + SDF_BUCKET_NUM * SDF_ENTRY_NUM_PER_BUCKET + exlOffset] = 1; //make child visible
		}

		break;
	}
}

__global__ void reAllocateSwappedOutVoxelBlocks_device(int *voxelAllocationList, ITMHashEntry *hashTable, int noTotalEntries,
	int *noAllocatedVoxelEntries, uchar *entriesVisibleType)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	int vbaIdx;
	ITMHashEntry hashEntry = hashTable[targetIdx];

	if (entriesVisibleType[targetIdx] > 0 && hashEntry.ptr == -1) //it is visible and has been previously allocated inside the hash, but deallocated from VBA
	{
		vbaIdx = atomicSub(&noAllocatedVoxelEntries[0], 1);
		if (vbaIdx >= 0) hashTable[targetIdx].ptr = voxelAllocationList[vbaIdx];
	}
}

template<bool useSwapping>
__global__ void buildVisibleList_device(ITMHashEntry *hashTable, ITMHashCacheState *cacheStates, int noTotalEntries, 
	int *liveEntryIDs, int *noLiveEntries, uchar *entriesVisibleType, Matrix4f M_d, Vector4f projParams_d, Vector2i imgSize, float voxelSize, int offsetToAdd)
{
	int targetIdx = threadIdx.x + blockIdx.x * blockDim.x;
	if (targetIdx > noTotalEntries - 1) return;

	__shared__ bool shouldPrefix;
	shouldPrefix = false;
	__syncthreads();

	unsigned char hashVisibleType = entriesVisibleType[targetIdx];
	const ITMHashEntry &hashEntry = hashTable[targetIdx];

	if (hashVisibleType > 0) shouldPrefix = true;
	else if (hashEntry.ptr >= -1)
	{
		bool isVisibleEnlarged = false;
		checkBlockVisibility<useSwapping>(hashVisibleType, isVisibleEnlarged, hashEntry.pos, M_d, projParams_d, voxelSize, imgSize);
		if (useSwapping) entriesVisibleType[targetIdx] = isVisibleEnlarged;
	}

	if (hashVisibleType > 0) shouldPrefix = true;

	if (useSwapping)
	{
		if (entriesVisibleType[targetIdx] > 0 && cacheStates[targetIdx].cacheFromHost != 2) cacheStates[targetIdx].cacheFromHost = 1;
	}

	__syncthreads();

	if (shouldPrefix)
	{
		int offset = computePrefixSum_device<int>(hashVisibleType > 0, noLiveEntries, blockDim.x * blockDim.y, threadIdx.x);
		if (offset != -1) liveEntryIDs[offset] = targetIdx + offsetToAdd;
	}
}

template class ITMLib::Engine::ITMSceneReconstructionEngine_CUDA<ITMVoxel, ITMVoxelIndex>;

