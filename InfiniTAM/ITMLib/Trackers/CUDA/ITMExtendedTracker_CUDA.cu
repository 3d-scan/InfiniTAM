#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMExtendedTracker_CUDA.h"
#include "../../Utils/ITMCUDAUtils.h"
#include "../Shared/ITMExtendedTracker_Shared.h"
#include "../../../ORUtils/CUDADefines.h"
#include <iostream>

using namespace ITMLib;

struct ITMExtendedTracker_CUDA::AccuCell {
	int numPoints;
	float f;
	float g[6];
	float h[6+5+4+3+2+1];
};

struct ITMExtendedTracker_KernelParameters_Depth {
	ITMExtendedTracker_CUDA::AccuCell *accu;
	float *depth;
	Matrix4f approxInvPose;
	Vector4f *pointsMap;
	Vector4f *normalsMap;
	Vector4f sceneIntrinsics;
	Vector2i sceneImageSize;
	Matrix4f scenePose;
	Vector4f viewIntrinsics;
	Vector2i viewImageSize;
	float spaceThresh;
	float viewFrustum_min, viewFrustum_max;
	int tukeyCutOff, framesToSkip, framesToWeight;
};

struct ITMExtendedTracker_KernelParameters_RGB {
	ITMExtendedTracker_CUDA::AccuCell *accu;
	Vector4f *pointsMap;
	const float *depths_curr;
	Vector2f *gradients;
	float *intensities_curr;
	float *intensities_prev;
	Vector2i viewImageSize;
	Vector2i sceneImageSize;
	Matrix4f approxInvPose;
	Matrix4f approxPose;
	Matrix4f scenePose;
	Vector4f projParams;
	float colourThresh;
	float viewFrustum_min, viewFrustum_max;
	float tukeyCutOff, framesToSkip, framesToWeight;
};

template<bool shortIteration, bool rotationOnly, bool useWeights>
__global__ void exDepthTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_Depth para);

template<bool shortIteration, bool rotationOnly, bool useWeights>
__global__ void exRGBTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_RGB para);

__global__ void exRGBTrackerProjectPrevImage_device(float *out_rgb, const float *in_rgb, const Vector4f *in_points, Vector2i imageSize, Vector2i sceneSize, Vector4f intrinsics, Matrix4f scenePose);

// host methods

ITMExtendedTracker_CUDA::ITMExtendedTracker_CUDA(Vector2i imgSize_d, Vector2i imgSize_rgb, bool useDepth, bool useColour,
	float colourWeight, TrackerIterationType *trackingRegime, int noHierarchyLevels,
	float terminationThreshold, float failureDetectorThreshold, float viewFrustum_min, float viewFrustum_max, int tukeyCutOff, int framesToSkip, int framesToWeight,
	const ITMLowLevelEngine *lowLevelEngine)
	: ITMExtendedTracker(imgSize_d, imgSize_rgb, useDepth, useColour, colourWeight, trackingRegime, noHierarchyLevels, terminationThreshold, failureDetectorThreshold, viewFrustum_min, viewFrustum_max,
	tukeyCutOff, framesToSkip, framesToWeight, lowLevelEngine, MEMORYDEVICE_CUDA)
{
	ORcudaSafeCall(hipHostMalloc((void**)&accu_host, sizeof(AccuCell)));
	ORcudaSafeCall(hipMalloc((void**)&accu_device, sizeof(AccuCell)));
}

ITMExtendedTracker_CUDA::~ITMExtendedTracker_CUDA(void)
{
	ORcudaSafeCall(hipHostFree(accu_host));
	ORcudaSafeCall(hipFree(accu_device));
}

int ITMExtendedTracker_CUDA::ComputeGandH_Depth(float &f, float *nabla, float *hessian, Matrix4f approxInvPose)
{
	Vector2i sceneImageSize = sceneHierarchyLevel_Depth->pointsMap->noDims;
	Vector2i viewImageSize = viewHierarchyLevel_Depth->depth->noDims;

	if (iterationType == TRACKER_ITERATION_NONE) return 0;

	bool shortIteration = (iterationType == TRACKER_ITERATION_ROTATION) || (iterationType == TRACKER_ITERATION_TRANSLATION);

	int noPara = shortIteration ? 3 : 6;

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)viewImageSize.x / (float)blockSize.x), (int)ceil((float)viewImageSize.y / (float)blockSize.y));

	ORcudaSafeCall(hipMemset(accu_device, 0, sizeof(AccuCell)));

	struct ITMExtendedTracker_KernelParameters_Depth args;
	args.accu = accu_device;
	args.depth = viewHierarchyLevel_Depth->depth->GetData(MEMORYDEVICE_CUDA);
	args.approxInvPose = approxInvPose;
	args.pointsMap = sceneHierarchyLevel_Depth->pointsMap->GetData(MEMORYDEVICE_CUDA);
	args.normalsMap = sceneHierarchyLevel_Depth->normalsMap->GetData(MEMORYDEVICE_CUDA);
	args.sceneIntrinsics = sceneHierarchyLevel_Depth->intrinsics;
	args.sceneImageSize = sceneImageSize;
	args.scenePose = scenePose;
	args.viewIntrinsics = viewHierarchyLevel_Depth->intrinsics;
	args.viewImageSize = viewHierarchyLevel_Depth->depth->noDims;
	args.spaceThresh = spaceThresh[levelId];
	args.viewFrustum_min = viewFrustum_min;
	args.viewFrustum_max = viewFrustum_max;
	args.tukeyCutOff = tukeyCutOff;
	args.framesToSkip = framesToSkip;
	args.framesToWeight = framesToWeight;

	//printf("%f %f\n", viewFrustum_min, viewFrustum_max);

	if (currentFrameNo < 100)
	{
		switch (iterationType)
		{
		case TRACKER_ITERATION_ROTATION:
			exDepthTrackerOneLevel_g_rt_device<true, true, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_TRANSLATION:
			exDepthTrackerOneLevel_g_rt_device<true, false, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_BOTH:
			exDepthTrackerOneLevel_g_rt_device<false, false, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		default: break;
		}
	}
	else
	{
		switch (iterationType)
		{
		case TRACKER_ITERATION_ROTATION:
			exDepthTrackerOneLevel_g_rt_device<true, true, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_TRANSLATION:
			exDepthTrackerOneLevel_g_rt_device<true, false, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_BOTH:
			exDepthTrackerOneLevel_g_rt_device<false, false, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		default: break;
		}
	}

	ORcudaSafeCall(hipMemcpy(accu_host, accu_device, sizeof(AccuCell), hipMemcpyDeviceToHost));

	for (int r = 0, counter = 0; r < noPara; r++) for (int c = 0; c <= r; c++, counter++) hessian[r + c * 6] = accu_host->h[counter];
	for (int r = 0; r < noPara; ++r) for (int c = r + 1; c < noPara; c++) hessian[r + c * 6] = hessian[c + r * 6];

	memcpy(nabla, accu_host->g, noPara * sizeof(float));

	if (accu_host->numPoints > 100)
	{
		for (int i = 0; i < 6 * 6; ++i) hessian[i] = hessian[i] / accu_host->numPoints;
		for (int i = 0; i < 6; ++i) nabla[i] = nabla[i] / accu_host->numPoints;

		f = accu_host->f / accu_host->numPoints;
	}
	else
	{
		f = 1e5f;
	}

	return accu_host->numPoints;
}

int ITMExtendedTracker_CUDA::ComputeGandH_RGB(float &f, float *nabla, float *hessian, Matrix4f approxInvPose)
{
//	Vector2i sceneImageSize = sceneHierarchyLevel_RGB->pointsMap->noDims;
//	Vector2i viewImageSize = viewHierarchyLevel_Intensity->intensity_current->noDims;
//
//	sceneHierarchyLevel_RGB->pointsMap->UpdateHostFromDevice();
//	previousProjectedIntensityLevel->depth->UpdateHostFromDevice();
//	viewHierarchyLevel_Intensity->intensity_current->UpdateHostFromDevice();
//	viewHierarchyLevel_Intensity->gradients->UpdateHostFromDevice();
//
//	Vector4f *locations = sceneHierarchyLevel_RGB->pointsMap->GetData(MEMORYDEVICE_CPU);
//	float *rgb_model = previousProjectedIntensityLevel->depth->GetData(MEMORYDEVICE_CPU);
//	float *rgb_live = viewHierarchyLevel_Intensity->intensity_current->GetData(MEMORYDEVICE_CPU);
//	Vector2f *gradients = viewHierarchyLevel_Intensity->gradients->GetData(MEMORYDEVICE_CPU);
//
//	Vector4f projParams = viewHierarchyLevel_Intensity->intrinsics;
//
//	Matrix4f approxPose;
//	approxInvPose.inv(approxPose);
////	approxPose = depthToRGBTransform * approxPose;
////	approxPose = approxPose;
//
//	if (iterationType == TRACKER_ITERATION_NONE) return 0;
//
//	bool shortIteration = (iterationType == TRACKER_ITERATION_ROTATION) || (iterationType == TRACKER_ITERATION_TRANSLATION);
//
//	float sumHessian[6 * 6], sumNabla[6], sumF;
//	int noValidPoints;
//	int noPara = shortIteration ? 3 : 6, noParaSQ = shortIteration ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;
//
//	noValidPoints = 0; sumF = 0.0f;
//	memset(sumHessian, 0, sizeof(float) * noParaSQ);
//	memset(sumNabla, 0, sizeof(float) * noPara);
//
//	float minF = 1e10, maxF = 0.f;
//	float minNabla[6], maxNabla[6];
//	float minHessian[noParaSQ], maxHessian[noParaSQ];
//
//	for(int i = 0; i < noPara; ++i)
//	{
//		minNabla[i] = 1e10f;
//		maxNabla[i] = -1e10f;
//	}
//
//	for(int i = 0; i < noParaSQ; ++i)
//	{
//		minHessian[i] = 1e10f;
//		maxHessian[i] = -1e10f;
//	}
//
//	for (int y = 0; y < sceneImageSize.y; y++) for (int x = 0; x < sceneImageSize.x; x++)
////	for (int y = 0; y < sceneImageSize.y; y++) for (int x = sceneImageSize.x - 1; x >= 0; x--)
////	for (int y = sceneImageSize.y - 1; y >= 0; y--) for (int x = sceneImageSize.x - 1; x >= 0; x--)
//	{
//		float localHessian[6 + 5 + 4 + 3 + 2 + 1], localNabla[6], localF = 0;
//
//		for (int i = 0; i < noPara; i++) localNabla[i] = 0.0f;
//		for (int i = 0; i < noParaSQ; i++) localHessian[i] = 0.0f;
//
//		bool isValidPoint = false;
//		float depthWeight = 1.f;
//
//		if (iterationType != TRACKER_ITERATION_TRANSLATION) // TODO translation not implemented yet
//		{
//			if (currentFrameNo < 100)
//				isValidPoint = computePerPointGH_exRGB_Ab<false>(localNabla, localF, localHessian, depthWeight,
//					locations[x + y * sceneImageSize.x], rgb_model[x + y * sceneImageSize.x], rgb_live, viewImageSize, x, y,
//					projParams, approxPose, approxInvPose, scenePose, gradients, colourThresh[levelId], viewFrustum_min, viewFrustum_max, tukeyCutOff, framesToSkip, framesToWeight, noPara);
//			else
//				isValidPoint = computePerPointGH_exRGB_Ab<true>(localNabla, localF, localHessian, depthWeight,
//					locations[x + y * sceneImageSize.x], rgb_model[x + y * sceneImageSize.x], rgb_live, viewImageSize, x, y,
//					projParams, approxPose, approxInvPose, scenePose, gradients, colourThresh[levelId], viewFrustum_min, viewFrustum_max, tukeyCutOff, framesToSkip, framesToWeight, noPara);
//		}
//
//		if (isValidPoint)
//		{
//			noValidPoints++;
//			sumF += localF;
//			for (int i = 0; i < noPara; i++) sumNabla[i] += localNabla[i];
//			for (int i = 0; i < noParaSQ; i++) sumHessian[i] += localHessian[i];
//
////			std::cerr << localNabla[0] << " " << localNabla[1] << "\n";
//
//			if(localF != 0.f)
//			{
//				minF = MIN(minF, localF);
//				maxF = MAX(maxF, localF);
//			}
//
//
//			for (int i = 0; i < noPara; i++)
//			{
//				if(localNabla[i] != 0.f)
//				{
//					minNabla[i] = MIN(minNabla[i], fabs(localNabla[i]));
//					maxNabla[i] = MAX(maxNabla[i], fabs(localNabla[i]));
//				}
//			}
//
//			for (int i = 0; i < noParaSQ; i++)
//				if(localHessian[i] != 0.f)
//				{
//					minHessian[i] = MIN(minHessian[i], fabs(localHessian[i]));
//					maxHessian[i] = MAX(maxHessian[i], fabs(localHessian[i]));
//				}
//		}
//	}
//
//	printf("Min F: %g - Max F: %g\n", minF, maxF);
//	printf("Min Nabla: ");
//	for (int i = 0; i < noPara; i++)
//	{
//		printf("%g - ", minNabla[i]);
//	}
//	printf("\nMax Nabla: ");
//	for (int i = 0; i < noPara; i++)
//	{
//		printf("%g - ", maxNabla[i]);
//	}
//	printf("\n");
//	printf("Min Hessian: ");
//	for (int i = 0; i < noParaSQ; i++)
//	{
//		printf("%g - ", minHessian[i]);
//	}
//	printf("\nMax Hessian: ");
//	for (int i = 0; i < noParaSQ; i++)
//	{
//		printf("%g - ", maxHessian[i]);
//	}
//	printf("\n\n");
//
//	for (int r = 0, counter = 0; r < noPara; r++) for (int c = 0; c <= r; c++, counter++) hessian[r + c * 6] = sumHessian[counter];
//	for (int r = 0; r < noPara; ++r) for (int c = r + 1; c < noPara; c++) hessian[r + c * 6] = hessian[c + r * 6];
//
////	memcpy(nabla, sumNabla, noPara * sizeof(float));
//	for (int r = 0; r < noPara; r++) nabla[r] = sumNabla[r];
//
//	if (noValidPoints > 100)
//	{
//		for (int i = 0; i < 6 * 6; ++i) hessian[i] = hessian[i] / noValidPoints;
//		for (int i = 0; i < 6; ++i) nabla[i] = nabla[i] / noValidPoints;
//
//		f = sumF / noValidPoints;
//	}
//	else
//	{
//		f = 1e25f;
//	}
//
//	return noValidPoints;

	Vector2i sceneImageSize = sceneHierarchyLevel_RGB->pointsMap->noDims;
	Vector2i viewImageSize = viewHierarchyLevel_Intensity->intensity_current->noDims;

	if (iterationType == TRACKER_ITERATION_NONE) return 0;

	Matrix4f approxPose;
	approxInvPose.inv(approxPose);
	approxPose = depthToRGBTransform * approxPose;

	bool shortIteration = (iterationType == TRACKER_ITERATION_ROTATION) || (iterationType == TRACKER_ITERATION_TRANSLATION);

	int noPara = shortIteration ? 3 : 6;

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)sceneImageSize.x / (float)blockSize.x), (int)ceil((float)sceneImageSize.y / (float)blockSize.y));

	ORcudaSafeCall(hipMemset(accu_device, 0, sizeof(AccuCell)));

	struct ITMExtendedTracker_KernelParameters_RGB args;
	args.accu = accu_device;
	args.depths_curr = viewHierarchyLevel_Depth->depth->GetData(MEMORYDEVICE_CUDA);
	args.intensities_curr = viewHierarchyLevel_Intensity->intensity_current->GetData(MEMORYDEVICE_CUDA);
	args.intensities_prev = viewHierarchyLevel_Intensity->intensity_prev->GetData(MEMORYDEVICE_CUDA);
	args.gradients = viewHierarchyLevel_Intensity->gradients->GetData(MEMORYDEVICE_CUDA);
	args.pointsMap = sceneHierarchyLevel_RGB->pointsMap->GetData(MEMORYDEVICE_CUDA);
	args.viewImageSize = viewImageSize;
	args.sceneImageSize = sceneImageSize;
	args.approxInvPose = approxInvPose;
	args.approxPose = approxPose;
	args.scenePose = scenePose;
	args.projParams = viewHierarchyLevel_Intensity->intrinsics;
	args.colourThresh = colourThresh[levelId];
	args.viewFrustum_min = viewFrustum_min;
	args.viewFrustum_max = viewFrustum_max;
	args.tukeyCutOff = tukeyCutOff;
	args.framesToSkip = framesToSkip;
	args.framesToWeight = framesToWeight;

	if (currentFrameNo < 100)
	{
		switch (iterationType)
		{
		case TRACKER_ITERATION_ROTATION:
			exRGBTrackerOneLevel_g_rt_device<true, true, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_TRANSLATION:
			exRGBTrackerOneLevel_g_rt_device<true, false, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_BOTH:
			exRGBTrackerOneLevel_g_rt_device<false, false, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		default: break;
		}
	}
	else
	{
		switch (iterationType)
		{
		case TRACKER_ITERATION_ROTATION:
			exRGBTrackerOneLevel_g_rt_device<true, true, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_TRANSLATION:
			exRGBTrackerOneLevel_g_rt_device<true, false, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_BOTH:
			exRGBTrackerOneLevel_g_rt_device<false, false, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		default: break;
		}
	}

	ORcudaSafeCall(hipMemcpy(accu_host, accu_device, sizeof(AccuCell), hipMemcpyDeviceToHost));

	for (int r = 0, counter = 0; r < noPara; r++) for (int c = 0; c <= r; c++, counter++) hessian[r + c * 6] = accu_host->h[counter];
	for (int r = 0; r < noPara; ++r) for (int c = r + 1; c < noPara; c++) hessian[r + c * 6] = hessian[c + r * 6];

	memcpy(nabla, accu_host->g, noPara * sizeof(float));

	if (accu_host->numPoints > 100)
	{
		for (int i = 0; i < 6 * 6; ++i) hessian[i] = hessian[i] / accu_host->numPoints;
		for (int i = 0; i < 6; ++i) nabla[i] = nabla[i] / accu_host->numPoints;

		f = accu_host->f / accu_host->numPoints;
	}
	else
	{
		f = 1e5f;
	}

	return accu_host->numPoints;
}

void ITMExtendedTracker_CUDA::ProjectPreviousRGBFrame(const Matrix4f &scenePose)
{
//	Vector2i imageSize = viewHierarchyLevel_Intensity->intensity_prev->noDims;
//	Vector2i sceneSize = sceneHierarchyLevel_RGB->pointsMap->noDims; // Also the size of the projected image
//
//	previousProjectedIntensityLevel->depth->ChangeDims(sceneSize);
//
//	sceneHierarchyLevel_RGB->pointsMap->UpdateHostFromDevice();
//	viewHierarchyLevel_Intensity->intensity_prev->UpdateHostFromDevice();
//	previousProjectedIntensityLevel->depth->UpdateHostFromDevice();
//
//	Vector4f projParams = viewHierarchyLevel_Intensity->intrinsics;
//	const Vector4f *pointsMap = sceneHierarchyLevel_RGB->pointsMap->GetData(MEMORYDEVICE_CPU);
//	const float *rgbIn = viewHierarchyLevel_Intensity->intensity_prev->GetData(MEMORYDEVICE_CPU);
//	float *rgbOut = previousProjectedIntensityLevel->depth->GetData(MEMORYDEVICE_CPU);
//
//	for (int y = 0; y < sceneSize.y; y++) for (int x = 0; x < sceneSize.x; x++)
//	{
//		projectPreviousPoint_exRGB(x, y, rgbOut, rgbIn, pointsMap, imageSize, sceneSize, projParams, scenePose);
//	}
//
//	sceneHierarchyLevel_RGB->pointsMap->UpdateDeviceFromHost();
//	viewHierarchyLevel_Intensity->intensity_prev->UpdateDeviceFromHost();
//	previousProjectedIntensityLevel->depth->UpdateDeviceFromHost();

	Vector2i imageSize = viewHierarchyLevel_Intensity->intensity_prev->noDims;
	Vector2i sceneSize = sceneHierarchyLevel_RGB->pointsMap->noDims; // Also the size of the projected image

	previousProjectedIntensityLevel->depth->ChangeDims(sceneSize); // Actual reallocation should happen only once per run.

	Vector4f projParams = viewHierarchyLevel_Intensity->intrinsics;
	const Vector4f *pointsMap = sceneHierarchyLevel_RGB->pointsMap->GetData(MEMORYDEVICE_CUDA);
	const float *rgbIn = viewHierarchyLevel_Intensity->intensity_prev->GetData(MEMORYDEVICE_CUDA);
	float *rgbOut = previousProjectedIntensityLevel->depth->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)sceneSize.x / (float)blockSize.x), (int)ceil((float)sceneSize.y / (float)blockSize.y));

	exRGBTrackerProjectPrevImage_device<<<gridSize, blockSize>>>(rgbOut, rgbIn, pointsMap, imageSize, sceneSize, projParams, scenePose);
	ORcudaKernelCheck;
}

// device functions
template<bool shortIteration, bool rotationOnly, bool useWeights>
__device__ void exDepthTrackerOneLevel_g_rt_device_main(ITMExtendedTracker_CUDA::AccuCell *accu, float *depth,
	Matrix4f approxInvPose, Vector4f *pointsMap, Vector4f *normalsMap, Vector4f sceneIntrinsics, Vector2i sceneImageSize, Matrix4f scenePose,
	Vector4f viewIntrinsics, Vector2i viewImageSize, float spaceThresh, float viewFrustum_min, float viewFrustum_max,
	int tukeyCutOff, int framesToSkip, int framesToWeight)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	int locId_local = threadIdx.x + threadIdx.y * blockDim.x;

	__shared__ float dim_shared1[256];
	__shared__ float dim_shared2[256];
	__shared__ float dim_shared3[256];
	__shared__ bool should_prefix;

	should_prefix = false;
	__syncthreads();

	const int noPara = shortIteration ? 3 : 6;
	const int noParaSQ = shortIteration ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;
	float A[noPara]; float b; float depthWeight = 1.0f;

	bool isValidPoint = false;

	if (x < viewImageSize.x && y < viewImageSize.y)
	{
		isValidPoint = computePerPointGH_exDepth_Ab<shortIteration, rotationOnly, useWeights>(A, b, x, y, depth[x + y * viewImageSize.x], depthWeight,
			viewImageSize, viewIntrinsics, sceneImageSize, sceneIntrinsics, approxInvPose, scenePose, pointsMap, normalsMap, spaceThresh,
			viewFrustum_min, viewFrustum_max, tukeyCutOff, framesToSkip, framesToWeight);

		if (isValidPoint) should_prefix = true;
	}

	if (!isValidPoint) {
		for (int i = 0; i < noPara; i++) A[i] = 0.0f;
		b = 0.0f;
	}

	__syncthreads();

	if (!should_prefix) return;

	{ //reduction for noValidPoints
		dim_shared1[locId_local] = isValidPoint;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();
		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);

		if (locId_local == 0) atomicAdd(&(accu->numPoints), (int)dim_shared1[locId_local]);
	}

	__syncthreads();

	{ //reduction for energy function value
		dim_shared1[locId_local] = rho(b, spaceThresh) * depthWeight;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();
		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);

		if (locId_local == 0) atomicAdd(&(accu->f), dim_shared1[locId_local]);
	}

	__syncthreads();

	//reduction for nabla
	for (unsigned char paraId = 0; paraId < noPara; paraId+=3)
	{
		dim_shared1[locId_local] = rho_deriv(b, spaceThresh) * depthWeight * A[paraId + 0];
		dim_shared2[locId_local] = rho_deriv(b, spaceThresh) * depthWeight * A[paraId + 1];
		dim_shared3[locId_local] = rho_deriv(b, spaceThresh) * depthWeight * A[paraId + 2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->g[paraId+0]), dim_shared1[0]);
			atomicAdd(&(accu->g[paraId+1]), dim_shared2[0]);
			atomicAdd(&(accu->g[paraId+2]), dim_shared3[0]);
		}
	}

	__syncthreads();

	float localHessian[noParaSQ];
#if (defined(__HIPCC__) && defined(__CUDA_ARCH__)) || (defined(__METALC__))
#pragma unroll
#endif
	for (unsigned char r = 0, counter = 0; r < noPara; r++)
	{
#if (defined(__HIPCC__) && defined(__CUDA_ARCH__)) || (defined(__METALC__))
#pragma unroll
#endif
		for (int c = 0; c <= r; c++, counter++) localHessian[counter] = rho_deriv2(b, spaceThresh) * depthWeight * A[r] * A[c];
	}

	//reduction for hessian
	for (unsigned char paraId = 0; paraId < noParaSQ; paraId+=3)
	{
		dim_shared1[locId_local] = localHessian[paraId+0];
		dim_shared2[locId_local] = localHessian[paraId+1];
		dim_shared3[locId_local] = localHessian[paraId+2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->h[paraId+0]), dim_shared1[0]);
			atomicAdd(&(accu->h[paraId+1]), dim_shared2[0]);
			atomicAdd(&(accu->h[paraId+2]), dim_shared3[0]);
		}
	}
}

template<bool shortIteration, bool rotationOnly, bool useWeights>
__device__ void exRGBTrackerOneLevel_g_rt_device_main(ITMExtendedTracker_CUDA::AccuCell *accu, 
	const float *depths_curr, const float *intensities_prev, const Vector2f *gradients, const float *intensities_curr,
	Matrix4f approxPose, Matrix4f approxInvPose, Matrix4f scenePose, Vector4f projParams,
	Vector2i imgSize, Vector2i sceneSize, float colourThresh, float viewFrustum_min, float viewFrustum_max,
	float tukeyCutoff, float framesToSkip, float framesToWeight)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	int locId_local = threadIdx.x + threadIdx.y * blockDim.x;

	__shared__ float dim_shared1[256];
	__shared__ float dim_shared2[256];
	__shared__ float dim_shared3[256];
	__shared__ bool should_prefix;

	should_prefix = false;
	__syncthreads();

	const int noPara = shortIteration ? 3 : 6;
	const int noParaSQ = shortIteration ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;
	float localHessian[noParaSQ];
	float localNabla[noPara];
	float localF;
	float depthWeight = 1.0f;

	bool isValidPoint = false;

	if (x < sceneSize.x && y < sceneSize.y)
	{
		// FIXME Translation only not implemented yet
		if(!shortIteration || rotationOnly)
		{
//			isValidPoint = computePerPointGH_exRGB_Ab<useWeights>(localNabla, localF, localHessian, depthWeight,
//				locations[x + y * sceneSize.x], intensity_model[x + y * sceneSize.x], intensity_live, imgSize, x, y,
//				projParams, approxPose, approxInvPose, scenePose, gradients, colourThresh, viewFrustum_min, viewFrustum_max, tukeyCutoff, framesToSkip, framesToWeight, noPara);

			isValidPoint = computePerPointGH_exRGB_inv_Ab<false>(
					localF,
					localNabla,
					localHessian,
					depthWeight,
					x,
					y,
					depths_curr,
					intensities_curr,
					intensities_prev,
					gradients,
					imgSize,
					imgSize,
					projParams,
					projParams,
					approxPose,
					approxInvPose,
					scenePose,
					colourThresh,
					viewFrustum_min,
					viewFrustum_max,
					tukeyCutoff,
					framesToSkip,
					framesToWeight,
					noPara
					);
		}

		if (isValidPoint) should_prefix = true;
	}

	if (!isValidPoint)
	{
		for (int i = 0; i < noParaSQ; i++) localHessian[i] = 0.0f;
		for (int i = 0; i < noPara; i++) localNabla[i] = 0.0f;
		localF = 0.0f;
	}

	__syncthreads();

	if (!should_prefix) return;

	{ //reduction for noValidPoints
		dim_shared1[locId_local] = isValidPoint;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();

		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);

		if (locId_local == 0) atomicAdd(&(accu->numPoints), (int)dim_shared1[locId_local]);
	}

	__syncthreads();

	{ //reduction for energy function value
		dim_shared1[locId_local] = localF;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();
		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);
		__syncthreads();

		if (locId_local == 0) atomicAdd(&(accu->f), dim_shared1[locId_local]);
	}

	__syncthreads();

	//reduction for nabla
	for (unsigned char paraId = 0; paraId < noPara; paraId += 3)
	{
		dim_shared1[locId_local] = localNabla[paraId + 0];
		dim_shared2[locId_local] = localNabla[paraId + 1];
		dim_shared3[locId_local] = localNabla[paraId + 2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->g[paraId + 0]), dim_shared1[0]);
			atomicAdd(&(accu->g[paraId + 1]), dim_shared2[0]);
			atomicAdd(&(accu->g[paraId + 2]), dim_shared3[0]);
		}
	}

	__syncthreads();

	//reduction for hessian
	for (unsigned char paraId = 0; paraId < noParaSQ; paraId += 3)
	{
		dim_shared1[locId_local] = localHessian[paraId + 0];
		dim_shared2[locId_local] = localHessian[paraId + 1];
		dim_shared3[locId_local] = localHessian[paraId + 2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->h[paraId + 0]), dim_shared1[0]);
			atomicAdd(&(accu->h[paraId + 1]), dim_shared2[0]);
			atomicAdd(&(accu->h[paraId + 2]), dim_shared3[0]);
		}
	}
}

template<bool shortIteration, bool rotationOnly, bool useWeights>
__global__ void exDepthTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_Depth para)
{
	exDepthTrackerOneLevel_g_rt_device_main<shortIteration, rotationOnly, useWeights>(para.accu, para.depth,
		para.approxInvPose, para.pointsMap, para.normalsMap, para.sceneIntrinsics, para.sceneImageSize, para.scenePose,
		para.viewIntrinsics, para.viewImageSize, para.spaceThresh, para.viewFrustum_min, para.viewFrustum_max,
		para.tukeyCutOff, para.framesToSkip, para.framesToWeight);
}

template<bool shortIteration, bool rotationOnly, bool useWeights>
__global__ void exRGBTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_RGB para)
{
	exRGBTrackerOneLevel_g_rt_device_main<shortIteration, rotationOnly, useWeights>(para.accu, para.depths_curr,
		para.intensities_prev, para.gradients, para.intensities_curr, para.approxPose, para.approxInvPose, para.scenePose,
		para.projParams, para.viewImageSize, para.sceneImageSize, para.colourThresh, para.viewFrustum_min, para.viewFrustum_max,
		para.tukeyCutOff, para.framesToSkip, para.framesToWeight);
}

__global__ void exRGBTrackerProjectPrevImage_device(float *out_rgb, const float *in_rgb, const Vector4f *in_points, Vector2i imageSize, Vector2i sceneSize, Vector4f intrinsics, Matrix4f scenePose)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	projectPreviousPoint_exRGB(x, y, out_rgb, in_rgb, in_points, imageSize, sceneSize, intrinsics, scenePose);
}
