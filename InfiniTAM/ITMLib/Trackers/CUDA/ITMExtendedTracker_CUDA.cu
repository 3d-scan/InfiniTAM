#include "hip/hip_runtime.h"
// Copyright 2014-2015 Isis Innovation Limited and the authors of InfiniTAM

#include "ITMExtendedTracker_CUDA.h"
#include "../../Utils/ITMCUDAUtils.h"
#include "../Shared/ITMExtendedTracker_Shared.h"
#include "../../../ORUtils/CUDADefines.h"
#include <iostream>

using namespace ITMLib;

struct ITMExtendedTracker_CUDA::AccuCell {
	int numPoints;
	float f;
	float g[6];
	float h[6+5+4+3+2+1];
};

struct ITMExtendedTracker_KernelParameters_Depth {
	ITMExtendedTracker_CUDA::AccuCell *accu;
	float *depth;
	Matrix4f approxInvPose;
	Vector4f *pointsMap;
	Vector4f *normalsMap;
	Vector4f sceneIntrinsics;
	Vector2i sceneImageSize;
	Matrix4f scenePose;
	Vector4f viewIntrinsics;
	Vector2i viewImageSize;
	float spaceThresh;
	float viewFrustum_min, viewFrustum_max;
	int tukeyCutOff, framesToSkip, framesToWeight;
};

struct ITMExtendedTracker_KernelParameters_RGB {
	ITMExtendedTracker_CUDA::AccuCell *accu;
	Vector4f *pointsMap;
	Vector4s *gx;
	Vector4s *gy;
	Vector4u *rgb_live;
	Vector4f *rgb_model;
	Vector2i viewImageSize;
	Vector2i sceneImageSize;
	Matrix4f approxInvPose;
	Matrix4f approxPose;
	Matrix4f scenePose;
	Vector4f projParams;
	float colourThresh;
	float viewFrustum_min, viewFrustum_max;
	float tukeyCutOff, framesToSkip, framesToWeight;
};

template<bool shortIteration, bool rotationOnly, bool useWeights>
__global__ void exDepthTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_Depth para);

template<bool shortIteration, bool rotationOnly, bool useWeights>
__global__ void exRGBTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_RGB para);

__global__ void exRGBTrackerProjectPrevImage_device(Vector4f *out_rgb, const Vector4u *in_rgb, const Vector4f *in_points, Vector2i imageSize, Vector2i sceneSize, Vector4f intrinsics, Matrix4f scenePose);

// host methods

ITMExtendedTracker_CUDA::ITMExtendedTracker_CUDA(Vector2i imgSize_d, Vector2i imgSize_rgb, bool useDepth, bool useColour,
	float colourWeight, TrackerIterationType *trackingRegime, int noHierarchyLevels,
	float terminationThreshold, float failureDetectorThreshold, float viewFrustum_min, float viewFrustum_max, int tukeyCutOff, int framesToSkip, int framesToWeight,
	const ITMLowLevelEngine *lowLevelEngine)
	: ITMExtendedTracker(imgSize_d, imgSize_rgb, useDepth, useColour, colourWeight, trackingRegime, noHierarchyLevels, terminationThreshold, failureDetectorThreshold, viewFrustum_min, viewFrustum_max,
	tukeyCutOff, framesToSkip, framesToWeight, lowLevelEngine, MEMORYDEVICE_CUDA)
{
	ORcudaSafeCall(hipHostMalloc((void**)&accu_host, sizeof(AccuCell)));
	ORcudaSafeCall(hipMalloc((void**)&accu_device, sizeof(AccuCell)));
}

ITMExtendedTracker_CUDA::~ITMExtendedTracker_CUDA(void)
{
	ORcudaSafeCall(hipHostFree(accu_host));
	ORcudaSafeCall(hipFree(accu_device));
}

int ITMExtendedTracker_CUDA::ComputeGandH_Depth(float &f, float *nabla, float *hessian, Matrix4f approxInvPose)
{
	Vector2i sceneImageSize = sceneHierarchyLevel_Depth->pointsMap->noDims;
	Vector2i viewImageSize = viewHierarchyLevel_Depth->depth->noDims;

	if (iterationType == TRACKER_ITERATION_NONE) return 0;

	bool shortIteration = (iterationType == TRACKER_ITERATION_ROTATION) || (iterationType == TRACKER_ITERATION_TRANSLATION);

	int noPara = shortIteration ? 3 : 6;

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)viewImageSize.x / (float)blockSize.x), (int)ceil((float)viewImageSize.y / (float)blockSize.y));

	ORcudaSafeCall(hipMemset(accu_device, 0, sizeof(AccuCell)));

	struct ITMExtendedTracker_KernelParameters_Depth args;
	args.accu = accu_device;
	args.depth = viewHierarchyLevel_Depth->depth->GetData(MEMORYDEVICE_CUDA);
	args.approxInvPose = approxInvPose;
	args.pointsMap = sceneHierarchyLevel_Depth->pointsMap->GetData(MEMORYDEVICE_CUDA);
	args.normalsMap = sceneHierarchyLevel_Depth->normalsMap->GetData(MEMORYDEVICE_CUDA);
	args.sceneIntrinsics = sceneHierarchyLevel_Depth->intrinsics;
	args.sceneImageSize = sceneImageSize;
	args.scenePose = scenePose;
	args.viewIntrinsics = viewHierarchyLevel_Depth->intrinsics;
	args.viewImageSize = viewHierarchyLevel_Depth->depth->noDims;
	args.spaceThresh = spaceThresh[levelId];
	args.viewFrustum_min = viewFrustum_min;
	args.viewFrustum_max = viewFrustum_max;
	args.tukeyCutOff = tukeyCutOff;
	args.framesToSkip = framesToSkip;
	args.framesToWeight = framesToWeight;

	//printf("%f %f\n", viewFrustum_min, viewFrustum_max);

	if (currentFrameNo < 100)
	{
		switch (iterationType)
		{
		case TRACKER_ITERATION_ROTATION:
			exDepthTrackerOneLevel_g_rt_device<true, true, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_TRANSLATION:
			exDepthTrackerOneLevel_g_rt_device<true, false, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_BOTH:
			exDepthTrackerOneLevel_g_rt_device<false, false, false> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		default: break;
		}
	}
	else
	{
		switch (iterationType)
		{
		case TRACKER_ITERATION_ROTATION:
			exDepthTrackerOneLevel_g_rt_device<true, true, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_TRANSLATION:
			exDepthTrackerOneLevel_g_rt_device<true, false, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		case TRACKER_ITERATION_BOTH:
			exDepthTrackerOneLevel_g_rt_device<false, false, true> << <gridSize, blockSize >> >(args);
			ORcudaKernelCheck;
			break;
		default: break;
		}
	}

	ORcudaSafeCall(hipMemcpy(accu_host, accu_device, sizeof(AccuCell), hipMemcpyDeviceToHost));

	for (int r = 0, counter = 0; r < noPara; r++) for (int c = 0; c <= r; c++, counter++) hessian[r + c * 6] = accu_host->h[counter];
	for (int r = 0; r < noPara; ++r) for (int c = r + 1; c < noPara; c++) hessian[r + c * 6] = hessian[c + r * 6];

	memcpy(nabla, accu_host->g, noPara * sizeof(float));

	if (accu_host->numPoints > 100)
	{
		for (int i = 0; i < 6 * 6; ++i) hessian[i] = hessian[i] / accu_host->numPoints;
		for (int i = 0; i < 6; ++i) nabla[i] = nabla[i] / accu_host->numPoints;

		f = accu_host->f / accu_host->numPoints;
	}
	else
	{
		f = 1e5f;
	}

	return accu_host->numPoints;
}

int ITMExtendedTracker_CUDA::ComputeGandH_RGB(float &f, float *nabla, float *hessian, Matrix4f approxInvPose)
{
	Vector2i sceneImageSize = sceneHierarchyLevel_RGB->pointsMap->noDims;
	Vector2i viewImageSize = viewHierarchyLevel_RGB->rgb_current->noDims;

	sceneHierarchyLevel_RGB->pointsMap->UpdateHostFromDevice();
	previousProjectedRGBLevel->depth->UpdateHostFromDevice();
	viewHierarchyLevel_RGB->rgb_current->UpdateHostFromDevice();
	viewHierarchyLevel_RGB->gX->UpdateHostFromDevice();
	viewHierarchyLevel_RGB->gY->UpdateHostFromDevice();

	Vector4f *locations = sceneHierarchyLevel_RGB->pointsMap->GetData(MEMORYDEVICE_CPU);
	Vector4f *rgb_model = previousProjectedRGBLevel->depth->GetData(MEMORYDEVICE_CPU);
	Vector4u *rgb_live = viewHierarchyLevel_RGB->rgb_current->GetData(MEMORYDEVICE_CPU);
	Vector4s *gx = viewHierarchyLevel_RGB->gX->GetData(MEMORYDEVICE_CPU);
	Vector4s *gy = viewHierarchyLevel_RGB->gY->GetData(MEMORYDEVICE_CPU);

	Vector4f projParams = viewHierarchyLevel_RGB->intrinsics;

	Matrix4f approxPose;
	approxInvPose.inv(approxPose);
//	approxPose = depthToRGBTransform * approxPose;
//	approxPose = approxPose;

	if (iterationType == TRACKER_ITERATION_NONE) return 0;

	bool shortIteration = (iterationType == TRACKER_ITERATION_ROTATION) || (iterationType == TRACKER_ITERATION_TRANSLATION);

	float sumHessian[6 * 6], sumNabla[6], sumF; int noValidPoints;
	int noPara = shortIteration ? 3 : 6, noParaSQ = shortIteration ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;

	noValidPoints = 0; sumF = 0.0f;
	memset(sumHessian, 0, sizeof(float) * noParaSQ);
	memset(sumNabla, 0, sizeof(float) * noPara);

	float minF = 1e10, maxF = 0.f;
	float minNabla[6], maxNabla[6];
	float minHessian[noParaSQ], maxHessian[noParaSQ];

	for(int i = 0; i < noPara; ++i)
	{
		minNabla[i] = 1e10f;
		maxNabla[i] = -1e10f;
	}

	for(int i = 0; i < noParaSQ; ++i)
	{
		minHessian[i] = 1e10f;
		maxHessian[i] = -1e10f;
	}

	for (int y = 0; y < viewImageSize.y; y++) for (int x = 0; x < viewImageSize.x; x++)
	{
		float localHessian[6 + 5 + 4 + 3 + 2 + 1], localNabla[6], localF = 0;

		for (int i = 0; i < noPara; i++) localNabla[i] = 0.0f;
		for (int i = 0; i < noParaSQ; i++) localHessian[i] = 0.0f;

		bool isValidPoint = false;
		float depthWeight = 1.f;

		if (iterationType != TRACKER_ITERATION_TRANSLATION) // TODO translation not implemented yet
		{
			if (currentFrameNo < 100)
				isValidPoint = computePerPointGH_exRGB_Ab<false>(localNabla, localF, localHessian, depthWeight,
					locations[x + y * sceneImageSize.x], rgb_model[x + y * sceneImageSize.x], rgb_live, viewImageSize, x, y,
					projParams, approxPose, approxInvPose, scenePose, gx, gy, colourThresh[levelId], viewFrustum_min, viewFrustum_max, tukeyCutOff, framesToSkip, framesToWeight, noPara);
			else
				isValidPoint = computePerPointGH_exRGB_Ab<true>(localNabla, localF, localHessian, depthWeight,
					locations[x + y * sceneImageSize.x], rgb_model[x + y * sceneImageSize.x], rgb_live, viewImageSize, x, y,
					projParams, approxPose, approxInvPose, scenePose, gx, gy, colourThresh[levelId], viewFrustum_min, viewFrustum_max, tukeyCutOff, framesToSkip, framesToWeight, noPara);
		}

		if (isValidPoint)
		{
			noValidPoints++;
			sumF += localF;
			for (int i = 0; i < noPara; i++) sumNabla[i] += localNabla[i];
			for (int i = 0; i < noParaSQ; i++) sumHessian[i] += localHessian[i];

			minF = MIN(minF, localF);
			maxF = MAX(maxF, localF);

			for (int i = 0; i < noPara; i++)
			{
				minNabla[i] = MIN(minNabla[i], localNabla[i]);
				maxNabla[i] = MAX(maxNabla[i], localNabla[i]);
			}

			for (int i = 0; i < noParaSQ; i++)
			{
				minHessian[i] = MIN(minHessian[i], localHessian[i]);
				maxHessian[i] = MAX(maxHessian[i], localHessian[i]);
			}
		}
	}

	printf("Min F: %g - Max F: %g\n", minF, maxF);
	printf("Min Nabla: ");
	for (int i = 0; i < noPara; i++)
	{
		printf("%g - ", minNabla[i]);
	}
	printf("\nMax Nabla: ");
	for (int i = 0; i < noPara; i++)
	{
		printf("%g - ", maxNabla[i]);
	}
	printf("\n");
	printf("Min Hessian: ");
	for (int i = 0; i < noParaSQ; i++)
	{
		printf("%g - ", minHessian[i]);
	}
	printf("\nMax Hessian: ");
	for (int i = 0; i < noParaSQ; i++)
	{
		printf("%g - ", maxHessian[i]);
	}
	printf("\n\n");

	for (int r = 0, counter = 0; r < noPara; r++) for (int c = 0; c <= r; c++, counter++) hessian[r + c * 6] = sumHessian[counter];
	for (int r = 0; r < noPara; ++r) for (int c = r + 1; c < noPara; c++) hessian[r + c * 6] = hessian[c + r * 6];

	memcpy(nabla, sumNabla, noPara * sizeof(float));

	if (noValidPoints > 100)
	{
		for (int i = 0; i < 6 * 6; ++i) hessian[i] = hessian[i] / noValidPoints;
		for (int i = 0; i < 6; ++i) nabla[i] = nabla[i] / noValidPoints;

		f = sumF / noValidPoints;
	}
	else
	{
		f = 1e5f;
	}

	return noValidPoints;

//	Vector2i sceneImageSize = sceneHierarchyLevel_RGB->pointsMap->noDims;
//	Vector2i viewImageSize = viewHierarchyLevel_RGB->rgb_current->noDims;
//
//	if (iterationType == TRACKER_ITERATION_NONE) return 0;
//
//	Matrix4f approxPose;
//	approxInvPose.inv(approxPose);
//	approxPose = depthToRGBTransform * approxPose;
//
//	bool shortIteration = (iterationType == TRACKER_ITERATION_ROTATION) || (iterationType == TRACKER_ITERATION_TRANSLATION);
//
//	int noPara = shortIteration ? 3 : 6;
//
//	dim3 blockSize(16, 16);
//	dim3 gridSize((int)ceil((float)sceneImageSize.x / (float)blockSize.x), (int)ceil((float)sceneImageSize.y / (float)blockSize.y));
//
//	ORcudaSafeCall(hipMemset(accu_device, 0, sizeof(AccuCell)));
//
//	struct ITMExtendedTracker_KernelParameters_RGB args;
//	args.accu = accu_device;
//	args.rgb_live = viewHierarchyLevel_RGB->rgb_current->GetData(MEMORYDEVICE_CUDA);
//	args.rgb_model = previousProjectedRGBLevel->depth->GetData(MEMORYDEVICE_CUDA);
//	args.gx = viewHierarchyLevel_RGB->gX->GetData(MEMORYDEVICE_CUDA);
//	args.gy = viewHierarchyLevel_RGB->gY->GetData(MEMORYDEVICE_CUDA);
//	args.pointsMap = sceneHierarchyLevel_RGB->pointsMap->GetData(MEMORYDEVICE_CUDA);
//	args.viewImageSize = viewImageSize;
//	args.sceneImageSize = sceneImageSize;
//	args.approxInvPose = approxInvPose;
//	args.approxPose = approxPose;
//	args.scenePose = scenePose;
//	args.projParams = viewHierarchyLevel_RGB->intrinsics;
//	args.colourThresh = colourThresh[levelId];
//	args.viewFrustum_min = viewFrustum_min;
//	args.viewFrustum_max = viewFrustum_max;
//	args.tukeyCutOff = tukeyCutOff;
//	args.framesToSkip = framesToSkip;
//	args.framesToWeight = framesToWeight;
//
//	if (currentFrameNo < 100)
//	{
//		switch (iterationType)
//		{
//		case TRACKER_ITERATION_ROTATION:
//			exRGBTrackerOneLevel_g_rt_device<true, true, false> << <gridSize, blockSize >> >(args);
//			ORcudaKernelCheck;
//			break;
//		case TRACKER_ITERATION_TRANSLATION:
//			exRGBTrackerOneLevel_g_rt_device<true, false, false> << <gridSize, blockSize >> >(args);
//			ORcudaKernelCheck;
//			break;
//		case TRACKER_ITERATION_BOTH:
//			exRGBTrackerOneLevel_g_rt_device<false, false, false> << <gridSize, blockSize >> >(args);
//			ORcudaKernelCheck;
//			break;
//		default: break;
//		}
//	}
//	else
//	{
//		switch (iterationType)
//		{
//		case TRACKER_ITERATION_ROTATION:
//			exRGBTrackerOneLevel_g_rt_device<true, true, true> << <gridSize, blockSize >> >(args);
//			ORcudaKernelCheck;
//			break;
//		case TRACKER_ITERATION_TRANSLATION:
//			exRGBTrackerOneLevel_g_rt_device<true, false, true> << <gridSize, blockSize >> >(args);
//			ORcudaKernelCheck;
//			break;
//		case TRACKER_ITERATION_BOTH:
//			exRGBTrackerOneLevel_g_rt_device<false, false, true> << <gridSize, blockSize >> >(args);
//			ORcudaKernelCheck;
//			break;
//		default: break;
//		}
//	}
//
//	ORcudaSafeCall(hipMemcpy(accu_host, accu_device, sizeof(AccuCell), hipMemcpyDeviceToHost));
//
//	for (int r = 0, counter = 0; r < noPara; r++) for (int c = 0; c <= r; c++, counter++) hessian[r + c * 6] = accu_host->h[counter];
//	for (int r = 0; r < noPara; ++r) for (int c = r + 1; c < noPara; c++) hessian[r + c * 6] = hessian[c + r * 6];
//
//	memcpy(nabla, accu_host->g, noPara * sizeof(float));
//
//	if (accu_host->numPoints > 100)
//	{
//		for (int i = 0; i < 6 * 6; ++i) hessian[i] = hessian[i] / accu_host->numPoints;
//		for (int i = 0; i < 6; ++i) nabla[i] = nabla[i] / accu_host->numPoints;
//
//		f = accu_host->f / accu_host->numPoints;
//	}
//	else
//	{
//		f = 1e5f;
//	}
//
//	return accu_host->numPoints;
}

void ITMExtendedTracker_CUDA::ProjectPreviousRGBFrame(const Matrix4f &scenePose)
{
//	Vector2i imageSize = viewHierarchyLevel_RGB->rgb_prev->noDims;
//	Vector2i sceneSize = sceneHierarchyLevel_RGB->pointsMap->noDims;
//
//	previousProjectedRGBLevel->depth->ChangeDims(sceneSize);
//
//	sceneHierarchyLevel_RGB->pointsMap->UpdateHostFromDevice();
//	viewHierarchyLevel_RGB->rgb_prev->UpdateHostFromDevice();
//	previousProjectedRGBLevel->depth->UpdateHostFromDevice();
//
//	Vector4f projParams = viewHierarchyLevel_RGB->intrinsics;
//	Vector4f *pointsMap = sceneHierarchyLevel_RGB->pointsMap->GetData(MEMORYDEVICE_CPU);
//	Vector4u *rgbIn = viewHierarchyLevel_RGB->rgb_prev->GetData(MEMORYDEVICE_CPU);
//	Vector4f *rgbOut = previousProjectedRGBLevel->depth->GetData(MEMORYDEVICE_CPU);
//
//	for (int y = 0; y < sceneSize.y; y++) for (int x = 0; x < sceneSize.x; x++)
//	{
//		projectPreviousPoint_exRGB(x, y, rgbOut, rgbIn, pointsMap, imageSize, sceneSize, projParams, scenePose);
//	}
//
//	sceneHierarchyLevel_RGB->pointsMap->UpdateDeviceFromHost();
//	viewHierarchyLevel_RGB->rgb_prev->UpdateDeviceFromHost();
//	previousProjectedRGBLevel->depth->UpdateDeviceFromHost();

	sceneHierarchyLevel_RGB->pointsMap->UpdateDeviceFromHost();
	viewHierarchyLevel_RGB->rgb_prev->UpdateDeviceFromHost();
	previousProjectedRGBLevel->depth->UpdateDeviceFromHost();

	Vector2i imageSize = viewHierarchyLevel_RGB->rgb_prev->noDims;
	Vector2i sceneSize = sceneHierarchyLevel_RGB->pointsMap->noDims; // Also the size of the projected image

	previousProjectedRGBLevel->depth->ChangeDims(sceneSize); // Actual reallocation should happen only once per run.

	Vector4f projParams = viewHierarchyLevel_RGB->intrinsics;
	const Vector4f *pointsMap = sceneHierarchyLevel_RGB->pointsMap->GetData(MEMORYDEVICE_CUDA);
	const Vector4u *rgbIn = viewHierarchyLevel_RGB->rgb_prev->GetData(MEMORYDEVICE_CUDA);
	Vector4f *rgbOut = previousProjectedRGBLevel->depth->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)sceneSize.x / (float)blockSize.x), (int)ceil((float)sceneSize.y / (float)blockSize.y));

	exRGBTrackerProjectPrevImage_device<<<gridSize, blockSize>>>(rgbOut, rgbIn, pointsMap, imageSize, sceneSize, projParams, scenePose);
	ORcudaKernelCheck;

	sceneHierarchyLevel_RGB->pointsMap->UpdateHostFromDevice();
	viewHierarchyLevel_RGB->rgb_prev->UpdateHostFromDevice();
	previousProjectedRGBLevel->depth->UpdateHostFromDevice();
}

// device functions
template<bool shortIteration, bool rotationOnly, bool useWeights>
__device__ void exDepthTrackerOneLevel_g_rt_device_main(ITMExtendedTracker_CUDA::AccuCell *accu, float *depth,
	Matrix4f approxInvPose, Vector4f *pointsMap, Vector4f *normalsMap, Vector4f sceneIntrinsics, Vector2i sceneImageSize, Matrix4f scenePose,
	Vector4f viewIntrinsics, Vector2i viewImageSize, float spaceThresh, float viewFrustum_min, float viewFrustum_max,
	int tukeyCutOff, int framesToSkip, int framesToWeight)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	int locId_local = threadIdx.x + threadIdx.y * blockDim.x;

	__shared__ float dim_shared1[256];
	__shared__ float dim_shared2[256];
	__shared__ float dim_shared3[256];
	__shared__ bool should_prefix;

	should_prefix = false;
	__syncthreads();

	const int noPara = shortIteration ? 3 : 6;
	const int noParaSQ = shortIteration ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;
	float A[noPara]; float b; float depthWeight = 1.0f;

	bool isValidPoint = false;

	if (x < viewImageSize.x && y < viewImageSize.y)
	{
		isValidPoint = computePerPointGH_exDepth_Ab<shortIteration, rotationOnly, useWeights>(A, b, x, y, depth[x + y * viewImageSize.x], depthWeight,
			viewImageSize, viewIntrinsics, sceneImageSize, sceneIntrinsics, approxInvPose, scenePose, pointsMap, normalsMap, spaceThresh,
			viewFrustum_min, viewFrustum_max, tukeyCutOff, framesToSkip, framesToWeight);

		if (isValidPoint) should_prefix = true;
	}

	if (!isValidPoint) {
		for (int i = 0; i < noPara; i++) A[i] = 0.0f;
		b = 0.0f;
	}

	__syncthreads();

	if (!should_prefix) return;

	{ //reduction for noValidPoints
		dim_shared1[locId_local] = isValidPoint;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();
		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);

		if (locId_local == 0) atomicAdd(&(accu->numPoints), (int)dim_shared1[locId_local]);
	}

	__syncthreads();

	{ //reduction for energy function value
		dim_shared1[locId_local] = rho(b, spaceThresh) * depthWeight;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();
		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);

		if (locId_local == 0) atomicAdd(&(accu->f), dim_shared1[locId_local]);
	}

	__syncthreads();

	//reduction for nabla
	for (unsigned char paraId = 0; paraId < noPara; paraId+=3)
	{
		dim_shared1[locId_local] = rho_deriv(b, spaceThresh) * depthWeight * A[paraId + 0];
		dim_shared2[locId_local] = rho_deriv(b, spaceThresh) * depthWeight * A[paraId + 1];
		dim_shared3[locId_local] = rho_deriv(b, spaceThresh) * depthWeight * A[paraId + 2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->g[paraId+0]), dim_shared1[0]);
			atomicAdd(&(accu->g[paraId+1]), dim_shared2[0]);
			atomicAdd(&(accu->g[paraId+2]), dim_shared3[0]);
		}
	}

	__syncthreads();

	float localHessian[noParaSQ];
#if (defined(__HIPCC__) && defined(__CUDA_ARCH__)) || (defined(__METALC__))
#pragma unroll
#endif
	for (unsigned char r = 0, counter = 0; r < noPara; r++)
	{
#if (defined(__HIPCC__) && defined(__CUDA_ARCH__)) || (defined(__METALC__))
#pragma unroll
#endif
		for (int c = 0; c <= r; c++, counter++) localHessian[counter] = rho_deriv2(b, spaceThresh) * depthWeight * A[r] * A[c];
	}

	//reduction for hessian
	for (unsigned char paraId = 0; paraId < noParaSQ; paraId+=3)
	{
		dim_shared1[locId_local] = localHessian[paraId+0];
		dim_shared2[locId_local] = localHessian[paraId+1];
		dim_shared3[locId_local] = localHessian[paraId+2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->h[paraId+0]), dim_shared1[0]);
			atomicAdd(&(accu->h[paraId+1]), dim_shared2[0]);
			atomicAdd(&(accu->h[paraId+2]), dim_shared3[0]);
		}
	}
}

template<bool shortIteration, bool rotationOnly, bool useWeights>
__device__ void exRGBTrackerOneLevel_g_rt_device_main(ITMExtendedTracker_CUDA::AccuCell *accu,
	Vector4f *locations, Vector4f *rgb_model, Vector4s *gx, Vector4s *gy, Vector4u *rgb,
	Matrix4f approxPose, Matrix4f approxInvPose, Matrix4f scenePose, Vector4f projParams,
	Vector2i imgSize, Vector2i sceneSize, float colourThresh, float viewFrustum_min, float viewFrustum_max,
	float tukeyCutoff, float framesToSkip, float framesToWeight)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	int locId_local = threadIdx.x + threadIdx.y * blockDim.x;

	__shared__ float dim_shared1[256];
	__shared__ float dim_shared2[256];
	__shared__ float dim_shared3[256];
	__shared__ bool should_prefix;

	should_prefix = false;
	__syncthreads();

	const int noPara = shortIteration ? 3 : 6;
	const int noParaSQ = shortIteration ? 3 + 2 + 1 : 6 + 5 + 4 + 3 + 2 + 1;
	float localHessian[noParaSQ];
	float A[noPara];
	float b;
	float depthWeight = 1.0f;

	bool isValidPoint = false;

	if (x < sceneSize.x && y < sceneSize.y)
	{
		// FIXME Translation only not implemented yet
		if(!shortIteration || rotationOnly)
		{
			isValidPoint = computePerPointGH_exRGB_Ab<useWeights>(A, b, localHessian, depthWeight, locations[x + y * sceneSize.x],
				rgb_model[x + y * sceneSize.x], rgb, imgSize, x, y,	projParams, approxPose, approxInvPose, scenePose, gx, gy,
				colourThresh, viewFrustum_min, viewFrustum_max, tukeyCutoff, framesToSkip, framesToWeight, noPara);
		}

		if (isValidPoint) should_prefix = true;
	}

	if (!isValidPoint)
	{
		for (int i = 0; i < noParaSQ; i++) localHessian[i] = 0.0f;
		for (int i = 0; i < noPara; i++) A[i] = 0.0f;
		b = 0.0f;
	}

	__syncthreads();

	if (!should_prefix) return;

	{ //reduction for noValidPoints
		dim_shared1[locId_local] = isValidPoint;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();

		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);

		if (locId_local == 0) atomicAdd(&(accu->numPoints), (int)dim_shared1[locId_local]);
	}

	__syncthreads();

	{ //reduction for energy function value
		dim_shared1[locId_local] = b;
		__syncthreads();

		if (locId_local < 128) dim_shared1[locId_local] += dim_shared1[locId_local + 128];
		__syncthreads();
		if (locId_local < 64) dim_shared1[locId_local] += dim_shared1[locId_local + 64];
		__syncthreads();

		if (locId_local < 32) warpReduce(dim_shared1, locId_local);
		__syncthreads();

		if (locId_local == 0) atomicAdd(&(accu->f), dim_shared1[locId_local]);
	}

	__syncthreads();

	//reduction for nabla
	for (unsigned char paraId = 0; paraId < noPara; paraId += 3)
	{
		dim_shared1[locId_local] = A[paraId + 0];
		dim_shared2[locId_local] = A[paraId + 1];
		dim_shared3[locId_local] = A[paraId + 2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->g[paraId + 0]), dim_shared1[0]);
			atomicAdd(&(accu->g[paraId + 1]), dim_shared2[0]);
			atomicAdd(&(accu->g[paraId + 2]), dim_shared3[0]);
		}
	}

	__syncthreads();

	//reduction for hessian
	for (unsigned char paraId = 0; paraId < noParaSQ; paraId += 3)
	{
		dim_shared1[locId_local] = localHessian[paraId + 0];
		dim_shared2[locId_local] = localHessian[paraId + 1];
		dim_shared3[locId_local] = localHessian[paraId + 2];
		__syncthreads();

		if (locId_local < 128) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 128];
			dim_shared2[locId_local] += dim_shared2[locId_local + 128];
			dim_shared3[locId_local] += dim_shared3[locId_local + 128];
		}
		__syncthreads();
		if (locId_local < 64) {
			dim_shared1[locId_local] += dim_shared1[locId_local + 64];
			dim_shared2[locId_local] += dim_shared2[locId_local + 64];
			dim_shared3[locId_local] += dim_shared3[locId_local + 64];
		}
		__syncthreads();

		if (locId_local < 32) {
			warpReduce(dim_shared1, locId_local);
			warpReduce(dim_shared2, locId_local);
			warpReduce(dim_shared3, locId_local);
		}
		__syncthreads();

		if (locId_local == 0) {
			atomicAdd(&(accu->h[paraId + 0]), dim_shared1[0]);
			atomicAdd(&(accu->h[paraId + 1]), dim_shared2[0]);
			atomicAdd(&(accu->h[paraId + 2]), dim_shared3[0]);
		}
	}
}

template<bool shortIteration, bool rotationOnly, bool useWeights>
__global__ void exDepthTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_Depth para)
{
	exDepthTrackerOneLevel_g_rt_device_main<shortIteration, rotationOnly, useWeights>(para.accu, para.depth,
		para.approxInvPose, para.pointsMap, para.normalsMap, para.sceneIntrinsics, para.sceneImageSize, para.scenePose,
		para.viewIntrinsics, para.viewImageSize, para.spaceThresh, para.viewFrustum_min, para.viewFrustum_max,
		para.tukeyCutOff, para.framesToSkip, para.framesToWeight);
}

template<bool shortIteration, bool rotationOnly, bool useWeights>
__global__ void exRGBTrackerOneLevel_g_rt_device(ITMExtendedTracker_KernelParameters_RGB para)
{
	exRGBTrackerOneLevel_g_rt_device_main<shortIteration, rotationOnly, useWeights>(para.accu, para.pointsMap,
		para.rgb_model, para.gx, para.gy, para.rgb_live, para.approxPose, para.approxInvPose, para.scenePose,
		para.projParams, para.viewImageSize, para.sceneImageSize, para.colourThresh, para.viewFrustum_min, para.viewFrustum_max,
		para.tukeyCutOff, para.framesToSkip, para.framesToWeight);
}

__global__ void exRGBTrackerProjectPrevImage_device(Vector4f *out_rgb, const Vector4u *in_rgb, const Vector4f *in_points, Vector2i imageSize, Vector2i sceneSize, Vector4f intrinsics, Matrix4f scenePose)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	projectPreviousPoint_exRGB(x, y, out_rgb, in_rgb, in_points, imageSize, sceneSize, intrinsics, scenePose);
}
